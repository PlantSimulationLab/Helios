#include "hip/hip_runtime.h"
/** \file "CollisionDetection.cu" CUDA source file for GPU-accelerated collision detection

    Copyright (C) 2016-2025 Brian Bailey

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, version 2.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <vector>

#include "helios_vector_types.h"

/**
 * \brief GPU-friendly BVH node structure
 *
 * This structure is optimized for GPU memory access patterns
 * and maintains 64-byte alignment for optimal performance.
 */
struct GPUBVHNode {
    float3 aabb_min; //!< Minimum corner of AABB (12 bytes)
    float3 aabb_max; //!< Maximum corner of AABB (12 bytes)
    unsigned int left_child; //!< Index of left child (0xFFFFFFFF if leaf) (4 bytes)
    unsigned int right_child; //!< Index of right child (0xFFFFFFFF if leaf) (4 bytes)
    unsigned int primitive_start; //!< Starting index in primitive array (4 bytes)
    unsigned int primitive_count; //!< Number of primitives (0 for internal nodes) (4 bytes)
    unsigned int is_leaf; //!< 1 if leaf, 0 if internal node (4 bytes)
    unsigned int padding; //!< Padding for alignment (4 bytes)
};

/**
 * \brief CUDA device function to test AABB intersection
 * \param[in] min1 Minimum corner of first AABB
 * \param[in] max1 Maximum corner of first AABB
 * \param[in] min2 Minimum corner of second AABB
 * \param[in] max2 Maximum corner of second AABB
 * \return True if AABBs intersect
 */
__device__ bool d_aabbIntersect(const float3 &min1, const float3 &max1, const float3 &min2, const float3 &max2) {
    return (min1.x <= max2.x && max1.x >= min2.x) && (min1.y <= max2.y && max1.y >= min2.y) && (min1.z <= max2.z && max1.z >= min2.z);
}

/**
 * \brief CUDA kernel for BVH traversal collision detection
 *
 * Each thread processes one query AABB and traverses the BVH to find collisions.
 *
 * \param[in] d_nodes Array of BVH nodes on GPU
 * \param[in] d_primitive_indices Array of primitive indices on GPU
 * \param[in] d_query_aabb_min Array of query AABB minimum corners
 * \param[in] d_query_aabb_max Array of query AABB maximum corners
 * \param[out] d_results Array to store collision results
 * \param[out] d_result_counts Array to store number of results per query
 * \param[in] num_queries Number of queries to process
 * \param[in] max_results_per_query Maximum results to store per query
 */
__global__ void bvhTraversalKernel(GPUBVHNode *d_nodes, unsigned int *d_primitive_indices, float3 *d_primitive_aabb_min, float3 *d_primitive_aabb_max, float3 *d_query_aabb_min, float3 *d_query_aabb_max, unsigned int *d_results,
                                   unsigned int *d_result_counts, int num_queries, int max_results_per_query) {

    int query_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (query_idx >= num_queries)
        return;

    float3 query_min = d_query_aabb_min[query_idx];
    float3 query_max = d_query_aabb_max[query_idx];

    unsigned int result_count = 0;
    unsigned int *query_results = &d_results[query_idx * max_results_per_query];

    // Stack-based traversal using shared memory for better performance
    __shared__ unsigned int node_stack[1024]; // Shared among threads in block
    int stack_size = 0;

    // Each thread gets its own portion of the shared stack
    int thread_stack_start = threadIdx.x * 32; // 32 entries per thread
    unsigned int *thread_stack = &node_stack[thread_stack_start];

    // Start traversal from root node
    thread_stack[0] = 0;
    stack_size = 1;

    while (stack_size > 0 && result_count < max_results_per_query) {

        // Pop node from stack
        stack_size--;
        unsigned int node_idx = thread_stack[stack_size];

        // Check if node index is valid
        if (node_idx == 0xFFFFFFFF)
            continue;

        GPUBVHNode node = d_nodes[node_idx];

        // Test if query AABB intersects node AABB
        if (!d_aabbIntersect(query_min, query_max, node.aabb_min, node.aabb_max)) {
            continue;
        }

        if (node.is_leaf) {
            // Check each primitive in this leaf individually
            for (unsigned int i = 0; i < node.primitive_count && result_count < max_results_per_query; i++) {
                unsigned int primitive_index = node.primitive_start + i;
                unsigned int primitive_id = d_primitive_indices[primitive_index];

                // Get primitive's AABB from pre-computed arrays (using array position, not UUID)
                float3 prim_min = d_primitive_aabb_min[primitive_index];
                float3 prim_max = d_primitive_aabb_max[primitive_index];

                // Only add to results if AABBs actually intersect
                if (d_aabbIntersect(query_min, query_max, prim_min, prim_max)) {
                    query_results[result_count] = primitive_id;
                    result_count++;
                }
            }
        } else {
            // Add child nodes to stack
            if (node.left_child != 0xFFFFFFFF && stack_size < 32) {
                thread_stack[stack_size] = node.left_child;
                stack_size++;
            }
            if (node.right_child != 0xFFFFFFFF && stack_size < 32) {
                thread_stack[stack_size] = node.right_child;
                stack_size++;
            }
        }
    }

    d_result_counts[query_idx] = result_count;
}

/**
 * \brief CUDA kernel to calculate ray path lengths through a regular voxel grid
 * 
 * This kernel computes the intersection of rays with voxels and accumulates
 * path length statistics for integration with aeriallidar and lidar plugins.
 * 
 * \param[in] num_rays Number of rays to process
 * \param[in] d_ray_origins Array of ray origin points
 * \param[in] d_ray_directions Array of ray direction vectors (normalized)
 * \param[in] grid_center Center of the voxel grid
 * \param[in] grid_size Total size of the voxel grid
 * \param[in] grid_divisions Number of divisions in x, y, z
 * \param[out] d_voxel_ray_counts Flattened array to accumulate ray counts per voxel
 * \param[out] d_voxel_path_lengths Flattened array to accumulate path lengths per voxel
 * \param[out] d_voxel_transmitted Flattened array to count transmitted rays per voxel
 */
__global__ void intersectRegularGridKernel(
    const size_t num_rays,
    float3 *d_ray_origins,
    float3 *d_ray_directions,
    float3 grid_center,
    float3 grid_size,
    int3 grid_divisions,
    int *d_voxel_ray_counts,
    float *d_voxel_path_lengths,
    int *d_voxel_transmitted
) {
    
    size_t ray_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ray_idx >= num_rays) {
        return;
    }
    
    float3 ray_origin = d_ray_origins[ray_idx];
    float3 ray_direction = d_ray_directions[ray_idx];
    
    // Calculate voxel size
    float3 voxel_size = make_float3(
        grid_size.x / static_cast<float>(grid_divisions.x),
        grid_size.y / static_cast<float>(grid_divisions.y),
        grid_size.z / static_cast<float>(grid_divisions.z)
    );
    
    float3 grid_min = make_float3(
        grid_center.x - 0.5f * grid_size.x,
        grid_center.y - 0.5f * grid_size.y,
        grid_center.z - 0.5f * grid_size.z
    );
    
    // Test intersection with each voxel in the grid
    for (int i = 0; i < grid_divisions.x; i++) {
        for (int j = 0; j < grid_divisions.y; j++) {
            for (int k = 0; k < grid_divisions.z; k++) {
                
                // Calculate voxel AABB
                float3 voxel_min = make_float3(
                    grid_min.x + i * voxel_size.x,
                    grid_min.y + j * voxel_size.y,
                    grid_min.z + k * voxel_size.z
                );
                
                float3 voxel_max = make_float3(
                    voxel_min.x + voxel_size.x,
                    voxel_min.y + voxel_size.y,
                    voxel_min.z + voxel_size.z
                );
                
                // Ray-AABB intersection test
                float t_min_x, t_max_x, t_min_y, t_max_y, t_min_z, t_max_z;
                
                // X slab
                float inv_dir_x = 1.0f / ray_direction.x;
                if (inv_dir_x >= 0) {
                    t_min_x = (voxel_min.x - ray_origin.x) * inv_dir_x;
                    t_max_x = (voxel_max.x - ray_origin.x) * inv_dir_x;
                } else {
                    t_min_x = (voxel_max.x - ray_origin.x) * inv_dir_x;
                    t_max_x = (voxel_min.x - ray_origin.x) * inv_dir_x;
                }
                
                // Y slab
                float inv_dir_y = 1.0f / ray_direction.y;
                if (inv_dir_y >= 0) {
                    t_min_y = (voxel_min.y - ray_origin.y) * inv_dir_y;
                    t_max_y = (voxel_max.y - ray_origin.y) * inv_dir_y;
                } else {
                    t_min_y = (voxel_max.y - ray_origin.y) * inv_dir_y;
                    t_max_y = (voxel_min.y - ray_origin.y) * inv_dir_y;
                }
                
                // Z slab
                float inv_dir_z = 1.0f / ray_direction.z;
                if (inv_dir_z >= 0) {
                    t_min_z = (voxel_min.z - ray_origin.z) * inv_dir_z;
                    t_max_z = (voxel_max.z - ray_origin.z) * inv_dir_z;
                } else {
                    t_min_z = (voxel_max.z - ray_origin.z) * inv_dir_z;
                    t_max_z = (voxel_min.z - ray_origin.z) * inv_dir_z;
                }
                
                // Find intersection parameters
                float t_enter = fmaxf(fmaxf(t_min_x, t_min_y), t_min_z);
                float t_exit = fminf(fminf(t_max_x, t_max_y), t_max_z);
                
                // Check if ray intersects voxel
                if (t_enter < t_exit && t_exit > 1e-6f) {
                    
                    // Calculate path length through voxel
                    float path_length = t_exit - t_enter;
                    
                    // Handle case where ray starts inside voxel
                    if (t_enter < 0) {
                        path_length = t_exit;
                    }
                    
                    // Calculate flattened voxel index
                    int voxel_idx = i * grid_divisions.y * grid_divisions.z + j * grid_divisions.z + k;
                    
                    // Accumulate statistics using atomic operations
                    atomicAdd(&d_voxel_ray_counts[voxel_idx], 1);
                    atomicAdd(&d_voxel_path_lengths[voxel_idx], path_length);
                    atomicAdd(&d_voxel_transmitted[voxel_idx], 1);
                }
            }
        }
    }
}

// C-style wrapper functions for calling from C++ code

extern "C" {

/**
 * \brief Launch BVH traversal kernel from C++ code
 *
 * \param[in] h_nodes Host array of BVH nodes
 * \param[in] node_count Number of BVH nodes
 * \param[in] h_primitive_indices Host array of primitive indices
 * \param[in] primitive_count Number of primitive indices
 * \param[in] h_query_aabb_min Host array of query AABB minimum corners
 * \param[in] h_query_aabb_max Host array of query AABB maximum corners
 * \param[in] num_queries Number of queries
 * \param[out] h_results Host array for results
 * \param[out] h_result_counts Host array for result counts
 * \param[in] max_results_per_query Maximum results per query
 */
void launchBVHTraversal(void *h_nodes, int node_count, unsigned int *h_primitive_indices, int primitive_count, float *h_primitive_aabb_min, float *h_primitive_aabb_max, float *h_query_aabb_min, float *h_query_aabb_max, int num_queries,
                        unsigned int *h_results, unsigned int *h_result_counts, int max_results_per_query) {

    if (num_queries == 0)
        return;

    // Allocate temporary GPU memory for query data and primitive AABBs
    float3 *d_query_min;
    float3 *d_query_max;
    float3 *d_primitive_min;
    float3 *d_primitive_max;
    unsigned int *d_results;
    unsigned int *d_result_counts;

    size_t query_size = num_queries * sizeof(float3);
    size_t primitive_aabb_size = primitive_count * sizeof(float3);
    size_t results_size = num_queries * max_results_per_query * sizeof(unsigned int);
    size_t counts_size = num_queries * sizeof(unsigned int);

    hipMalloc((void **) &d_query_min, query_size);
    hipMalloc((void **) &d_query_max, query_size);
    hipMalloc((void **) &d_primitive_min, primitive_aabb_size);
    hipMalloc((void **) &d_primitive_max, primitive_aabb_size);
    hipMalloc((void **) &d_results, results_size);
    hipMalloc((void **) &d_result_counts, counts_size);

    // Convert query data to float3 format
    std::vector<float3> query_min_vec(num_queries);
    std::vector<float3> query_max_vec(num_queries);
    for (int i = 0; i < num_queries; i++) {
        query_min_vec[i] = make_float3(h_query_aabb_min[i * 3], h_query_aabb_min[i * 3 + 1], h_query_aabb_min[i * 3 + 2]);
        query_max_vec[i] = make_float3(h_query_aabb_max[i * 3], h_query_aabb_max[i * 3 + 1], h_query_aabb_max[i * 3 + 2]);
    }

    // Convert primitive AABB data to float3 format
    std::vector<float3> primitive_min_vec(primitive_count);
    std::vector<float3> primitive_max_vec(primitive_count);
    for (int i = 0; i < primitive_count; i++) {
        primitive_min_vec[i] = make_float3(h_primitive_aabb_min[i * 3], h_primitive_aabb_min[i * 3 + 1], h_primitive_aabb_min[i * 3 + 2]);
        primitive_max_vec[i] = make_float3(h_primitive_aabb_max[i * 3], h_primitive_aabb_max[i * 3 + 1], h_primitive_aabb_max[i * 3 + 2]);
    }

    // Copy query and primitive AABB data to GPU
    hipMemcpy(d_query_min, query_min_vec.data(), query_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query_max, query_max_vec.data(), query_size, hipMemcpyHostToDevice);
    hipMemcpy(d_primitive_min, primitive_min_vec.data(), primitive_aabb_size, hipMemcpyHostToDevice);
    hipMemcpy(d_primitive_max, primitive_max_vec.data(), primitive_aabb_size, hipMemcpyHostToDevice);

    // Launch kernel
    int block_size = 256;
    int num_blocks = (num_queries + block_size - 1) / block_size;

    bvhTraversalKernel<<<num_blocks, block_size>>>((GPUBVHNode *) h_nodes, (unsigned int *) h_primitive_indices, d_primitive_min, d_primitive_max, d_query_min, d_query_max, d_results, d_result_counts, num_queries, max_results_per_query);

    hipDeviceSynchronize();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    // Copy results back
    hipMemcpy(h_results, d_results, results_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_result_counts, d_result_counts, counts_size, hipMemcpyDeviceToHost);

    // Clean up temporary memory
    hipFree(d_query_min);
    hipFree(d_query_max);
    hipFree(d_primitive_min);
    hipFree(d_primitive_max);
    hipFree(d_results);
    hipFree(d_result_counts);
}

/**
 * \brief Launch cone intersection kernel from C++ code
 *
 * \param[in] h_nodes Host array of BVH nodes
 * \param[in] node_count Number of BVH nodes
 * \param[in] h_primitive_indices Host array of primitive indices
 * \param[in] primitive_count Number of primitive indices
 * \param[in] h_cone_origins Host array of cone origins
 * \param[in] h_cone_directions Host array of cone directions
 * \param[in] h_cone_angles Host array of cone angles
 * \param[in] h_max_distances Host array of maximum distances
 * \param[in] num_queries Number of queries
 * \param[out] h_results Host array for results
 * \param[out] h_result_counts Host array for result counts
 * \param[in] max_results_per_query Maximum results per query
 */
void launchConeIntersection(void *h_nodes, int node_count, unsigned int *h_primitive_indices, int primitive_count, float *h_cone_origins, float *h_cone_directions, float *h_cone_angles, float *h_max_distances, int num_queries,
                            unsigned int *h_results, unsigned int *h_result_counts, int max_results_per_query) {

    // Stub implementation - will be completed in Phase 3
    for (int i = 0; i < num_queries; i++) {
        h_result_counts[i] = 0;
    }
}

/**
 * \brief Launch CUDA kernel for regular grid voxel ray path length calculation
 */
void launchVoxelRayPathLengths(
    int num_rays,
    float *h_ray_origins,
    float *h_ray_directions,
    float grid_center_x, float grid_center_y, float grid_center_z,
    float grid_size_x, float grid_size_y, float grid_size_z,
    int grid_divisions_x, int grid_divisions_y, int grid_divisions_z,
    int *h_voxel_ray_counts,
    float *h_voxel_path_lengths,
    int *h_voxel_transmitted
) {
    
    // Allocate device memory
    float3 *d_ray_origins, *d_ray_directions;
    int *d_voxel_ray_counts, *d_voxel_transmitted;
    float *d_voxel_path_lengths;
    
    size_t ray_data_size = num_rays * 3 * sizeof(float);
    size_t voxel_count = grid_divisions_x * grid_divisions_y * grid_divisions_z;
    size_t voxel_int_size = voxel_count * sizeof(int);
    size_t voxel_float_size = voxel_count * sizeof(float);
    
    // Allocate memory
    hipMalloc(&d_ray_origins, ray_data_size);
    hipMalloc(&d_ray_directions, ray_data_size);
    hipMalloc(&d_voxel_ray_counts, voxel_int_size);
    hipMalloc(&d_voxel_transmitted, voxel_int_size);
    hipMalloc(&d_voxel_path_lengths, voxel_float_size);
    
    // Copy input data to device
    hipMemcpy(d_ray_origins, h_ray_origins, ray_data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ray_directions, h_ray_directions, ray_data_size, hipMemcpyHostToDevice);
    hipMemset(d_voxel_ray_counts, 0, voxel_int_size);
    hipMemset(d_voxel_transmitted, 0, voxel_int_size);
    hipMemset(d_voxel_path_lengths, 0, voxel_float_size);
    
    // Launch kernel
    dim3 block_size(256);
    dim3 grid_size((num_rays + block_size.x - 1) / block_size.x);
    
    float3 grid_center = make_float3(grid_center_x, grid_center_y, grid_center_z);
    float3 grid_size_vec = make_float3(grid_size_x, grid_size_y, grid_size_z);
    int3 grid_divisions_vec = make_int3(grid_divisions_x, grid_divisions_y, grid_divisions_z);
    
    intersectRegularGridKernel<<<grid_size, block_size>>>(
        num_rays,
        d_ray_origins,
        d_ray_directions,
        grid_center,
        grid_size_vec,
        grid_divisions_vec,
        d_voxel_ray_counts,
        d_voxel_path_lengths,
        d_voxel_transmitted
    );
    
    hipDeviceSynchronize();
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA voxel kernel launch error: %s\n", hipGetErrorString(err));
    }
    
    // Copy results back to host
    hipMemcpy(h_voxel_ray_counts, d_voxel_ray_counts, voxel_int_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_voxel_path_lengths, d_voxel_path_lengths, voxel_float_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_voxel_transmitted, d_voxel_transmitted, voxel_int_size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_ray_origins);
    hipFree(d_ray_directions);
    hipFree(d_voxel_ray_counts);
    hipFree(d_voxel_transmitted);
    hipFree(d_voxel_path_lengths);
}

} // extern "C"
