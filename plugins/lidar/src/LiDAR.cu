#include "hip/hip_runtime.h"
/** \file "LiDAR.cu" CUDA-related LiDAR plug-in functions 
    \author Brian Bailey, Eric Kent

    Copyright (C) 2016-2022 Brian Bailey

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, version 2.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

*/

#include "LiDAR.cuh"
#include "LiDAR.h"

using namespace LIDAR_CUDA;

__host__ __device__ float2 LIDAR_CUDA::operator+(const float2 &a, const float2 &b) {

  return make_float2(a.x+b.x, a.y+b.y);

}

__host__ __device__ float3 LIDAR_CUDA::operator+(const float3 &a, const float3 &b) {

  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);

}

__host__ __device__ float2 LIDAR_CUDA::operator-(const float2 &a, const float2 &b) {

  return make_float2(a.x-b.x, a.y-b.y);

}

__host__ __device__ float3 LIDAR_CUDA::operator-(const float3 &a, const float3 &b) {

  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);

}

__host__ __device__ float LIDAR_CUDA::operator*(const float2 &a, const float2 &b) {

  return a.x*b.x + a.y*b.y;

}

__host__ __device__ float LIDAR_CUDA::operator*(const float3 &a, const float3 &b) {

  return a.x*b.x + a.y*b.y + a.z*b.z;

}

__host__ __device__ float3 LIDAR_CUDA::operator*(const float3 &a, const float& b) {
  float3 v = a;
  v.x *= b;
  v.y *= b;
  v.z *= b;
  return v;
}

__host__ __device__ float3 LIDAR_CUDA::operator/(const float3 &a, const float& b) {
  float3 v = a;
  v.x /= b;
  v.y /= b;
  v.z /= b;
  return v;
}

__host__ __device__ float3 LIDAR_CUDA::cross(const float3 &a, const float3 &b){
  float3 c;
  c.x=a.y*b.z - b.y*a.z;
  c.y=b.x*a.z - a.x*b.z;
  c.z=a.x*b.y - b.x*a.y;
  return c;
}

__host__ __device__ float LIDAR_CUDA::magnitude(const float3 &a){
  return sqrt( a.x*a.x + a.y*a.y + a.z*a.z );
}

__host__ __device__ float3 LIDAR_CUDA::normalize(const float3 &a){
  return a/magnitude(a);
}

float2 LIDAR_CUDA::vec2tofloat2( helios::vec2 v2 ){
  float2 f2;
  f2.x=v2.x;
  f2.y=v2.y;
  return f2;
}

helios::vec2 LIDAR_CUDA::float2tovec2( float2 f2 ){
  helios::vec2 v2;
  v2.x=f2.x;
  v2.y=f2.y;
  return v2;
}

float3 LIDAR_CUDA::vec3tofloat3( helios::vec3 v3 ){
  float3 f3;
  f3.x=v3.x;
  f3.y=v3.y;
  f3.z=v3.z;
  return f3;
}

helios::vec3 LIDAR_CUDA::float3tovec3( float3 f3 ){
  helios::vec3 v3;
  v3.x=f3.x;
  v3.y=f3.y;
  v3.z=f3.z;
  return v3;
}

__device__ float3 LIDAR_CUDA::d_rotatePoint(const float3 &position, const float &theta, const float &phi) {

  float Ry[3][3], Rz[3][3];

  float st = sinf(theta);
  float ct = cosf(theta);

  float sp = sinf(phi);
  float cp = cosf(phi);
	
  // Setup the rotation matrix, this matrix is based off of the rotation matrix used in glRotatef.
  Ry[0][0] = ct;
  Ry[0][1] = 0.f;
  Ry[0][2] = st;
  Ry[1][0] = 0.f;
  Ry[1][1] = 1.f;
  Ry[1][2] = 0.f;
  Ry[2][0] = -st;
  Ry[2][1] = 0.f;
  Ry[2][2] = ct;

  Rz[0][0] = cp;
  Rz[0][1] = -sp;
  Rz[0][2] = 0.f;
  Rz[1][0] = sp;
  Rz[1][1] = cp;
  Rz[1][2] = 0.f;
  Rz[2][0] = 0.f;
  Rz[2][1] = 0.f;
  Rz[2][2] = 1.f;

  // Multiply Ry*Rz

  float rotMat[3][3] = {0.f};

  for( int i=0;i<3;i++){
    for(int j=0;j<3;j++){
      for(int k=0;k<3;k++){
	rotMat[i][j]=rotMat[i][j]+Rz[i][k]*Ry[k][j];
      }
    }
  }
	
  // Multiply the rotation matrix with the position vector.
  float3 tmp;
  tmp.x = rotMat[0][0] * position.x + rotMat[0][1] * position.y + rotMat[0][2] * position.z;
  tmp.y = rotMat[1][0] * position.x + rotMat[1][1] * position.y + rotMat[1][2] * position.z;
  tmp.z = rotMat[2][0] * position.x + rotMat[2][1] * position.y + rotMat[2][2] * position.z;
  
  return tmp;
	
}

__device__ float3 LIDAR_CUDA::d_sphere2cart( float radius, float elevation, float azimuth ){

  float3 Cartesian;

  Cartesian.x = radius*cos(elevation)*sin(azimuth);
  Cartesian.y = radius*cos(elevation)*cos(azimuth);
  Cartesian.z = radius*sin(elevation);

  return Cartesian;
  
}

__host__ __device__ float LIDAR_CUDA::atan2_2pi( const float& y, const float& x){

  float v=0;

  if(x>0.f){
    v=atan(y/x);
  }
  if(y>=0.f && x<0.f){
    v=M_PI+atan(y/x);
  }
  if(y<0.f && x<0.f){
    v=-M_PI+atan(y/x);
  }
  if(y>0.f && x==0.f){
    v=0.5f*M_PI;
  }
  if(y<0.f && x==0.f){
    v=-0.5f*M_PI;
  }
  if(v<0.f){
    v=v+2.f*M_PI;
  }
  return v;

}

__host__ __device__ float LIDAR_CUDA::acos_safe( float x ){
  if (x < -1.0) x = -1.0 ;
  else if (x > 1.0) x = 1.0 ;
  return acosf(x) ;
}

__host__ int LIDAR_CUDA::randu( int imin, int imax  ){

  float ru = float(rand()) / float(RAND_MAX + 1.); 

  if( imin==imax || imin>imax ){
    return imin;
  }else{
    return imin + round(float(imax-imin)*ru);
  }
    
}

__device__ float2 LIDAR_CUDA::d_sampleDisk( float radius, uint seed ){

  // use Box-Muller tranform approach
  // the input radius is the place where the intensity decreases to 1/e^2 (~0.1353) of the intensity at the beam center
    
  // sample from uniform random distribution [0,1] for radial distance and angle  
  float U1 = rnd(seed);
  float U2 = rnd(seed);
  
  // calculate the standard deviation of a normal distribution where  intensity decreases to 1/e^2 at the input radius
  float sd = sqrt(0.25*radius*radius);
  
  // formula from https://www.alanzucconi.com/2015/09/16/how-to-sample-from-a-gaussian-distribution/
  float R = sqrt(-2.f*logf(U1))*sd;
  float angle = 2.f*M_PI*U2;
  
  float2 sp;
  //find x,y point on unit disk
  sp.x = R*cosf(angle);
  sp.y = R*sinf(angle);
  return sp;
  
}

bool LIDAR_CUDA::sortcol0( const std::vector<float>& v0, const std::vector<float>& v1 ){
  return v0.at(0)<v1.at(0);
}

// bool LIDAR_CUDA::sortcol1( const std::vector<float>& v0, const std::vector<float>& v1 ){
//   return v0.at(1)<v1.at(1);
// }

// bool LIDAR_CUDA::sortcol0( const std::vector<double>& v0, const std::vector<double>& v1 ){
//   return v0.at(0)<v1.at(0);
// }

bool LIDAR_CUDA::sortcol1( const std::vector<double>& v0, const std::vector<double>& v1 ){
  return v0.at(1)<v1.at(1);
}

void LiDARcloud::calculateHitGridCellGPU( void ){

  if( printmessages ){
    std::cout << "Grouping hit points by grid cell..." << std::flush;
  }
    
  // ---- Hit Points ---- //

  float3* d_hit_xyz;

  uint Nscans = getScanCount();

  std::size_t total_hits = getHitCount();

  const uint N = total_hits;
  
  float3* hit_xyz = (float3*)malloc(N * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_xyz,N*sizeof(float3)) ); //allocate device memory

  //copy scan data into the host buffer
  for( std::size_t r=0; r<getHitCount(); r++ ){
    hit_xyz[r] = vec3tofloat3(getHitXYZ(r));
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_hit_xyz, hit_xyz, N*sizeof(float3), hipMemcpyHostToDevice) );

  // ---- Grid Cells ---- //

  //! Grid cell centers on device (GPU) memory
  float3* d_grid_center;

  //! Grid cell global anchor on device (GPU) memory
  float3* d_grid_anchor;

  //! Grid sizes on device (GPU) memory
  float3* d_grid_size;

  //! Grid rotations on device (GPU) memory
  float* d_grid_rotation;

  const uint Ncells = getGridCellCount();

  float3* center = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_center,Ncells*sizeof(float3)) ); //allocate device memory

  float3* anchor = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_anchor,Ncells*sizeof(float3)) ); //allocate device memory

  float3* size = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_size,Ncells*sizeof(float3)) ); //allocate device memory

  float* rotation = (float*)malloc(Ncells * sizeof(float)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_rotation,Ncells*sizeof(float)) ); //allocate device memory
  
  //copy grid data into the host buffer
  for( int c=0; c<Ncells; c++ ){
    center[c] = vec3tofloat3(getCellCenter(c));
    anchor[c] = vec3tofloat3(getCellGlobalAnchor(c));
    size[c] = vec3tofloat3(getCellSize(c));
    rotation[c] = getCellRotation(c);
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_center, center, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_anchor, anchor, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_size, size, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_rotation, rotation, Ncells*sizeof(float), hipMemcpyHostToDevice) );

  free(hit_xyz);
  free(center);
  free(anchor);
  free(size);
  free(rotation);

  // Result buffer
  int* hit_vol = (int*)malloc(total_hits * sizeof(int));
  int* d_hit_vol;
  CUDA_CHECK_ERROR( hipMalloc(&d_hit_vol,total_hits*sizeof(int)) );

  dim3 dimBlock( 64, 1, 1 );
  dim3 dimGrid( ceil(total_hits/64.f), 1, 1 );
  if( total_hits==0 ){
    std::cout << "WARNING (calculateHitGridcellGPU): There are no hits currently in the point cloud. Skipping grid cell binning calculation." << std::endl;
  }else{
    insideVolume <<< dimGrid, dimBlock >>>( total_hits, d_hit_xyz, getGridCellCount(), d_grid_size, d_grid_center, d_grid_anchor, d_grid_rotation, d_hit_vol );
  }
    
  CUDA_CHECK_ERROR( hipPeekAtLastError() );
  CUDA_CHECK_ERROR( hipDeviceSynchronize() );
  
  CUDA_CHECK_ERROR( hipMemcpy(hit_vol, d_hit_vol, total_hits*sizeof(int), hipMemcpyDeviceToHost) );

  for( std::size_t r=0; r<getHitCount(); r++ ){
    setHitGridCell( r, hit_vol[r] );
  }

  free(hit_vol);

  CUDA_CHECK_ERROR( hipFree(d_hit_vol) );
  CUDA_CHECK_ERROR( hipFree(d_hit_xyz) );
  CUDA_CHECK_ERROR( hipFree(d_grid_center) );
  CUDA_CHECK_ERROR( hipFree(d_grid_anchor) );
  CUDA_CHECK_ERROR( hipFree(d_grid_size) );
  CUDA_CHECK_ERROR( hipFree(d_grid_rotation) );  

  hitgridcellcomputed = true;

  if( printmessages ){
    std::cout << "done." << std::endl;
  }
    
}

void LiDARcloud::sourcesInsideGridCellGPU( void ){

  if( printmessages ){
    std::cout << "Checking that scan origins are not inside grid cells..." << std::flush;
  }

  // ---- Hit Points ---- //

  float3* d_hit_xyz;

  uint Nscans = getScanCount();

  std::size_t total_hits = Nscans;

  const uint N = total_hits;
  
  float3* hit_xyz = (float3*)malloc(N * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_xyz,N*sizeof(float3)) ); //allocate device memory

  //copy scan data into the host buffer
  for( std::size_t s=0; s<Nscans; s++ ){
    hit_xyz[s] = vec3tofloat3(getScanOrigin(s));
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_hit_xyz, hit_xyz, N*sizeof(float3), hipMemcpyHostToDevice) );

  // ---- Grid Cells ---- //

  //! Grid cell centers on device (GPU) memory
  float3* d_grid_center;

  //! Grid cell global anchor on device (GPU) memory
  float3* d_grid_anchor;

  //! Grid sizes on device (GPU) memory
  float3* d_grid_size;

  //! Grid rotations on device (GPU) memory
  float* d_grid_rotation;

  const uint Ncells = getGridCellCount();

  float3* center = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_center,Ncells*sizeof(float3)) ); //allocate device memory

  float3* anchor = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_anchor,Ncells*sizeof(float3)) ); //allocate device memory

  float3* size = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_size,Ncells*sizeof(float3)) ); //allocate device memory

  float* rotation = (float*)malloc(Ncells * sizeof(float)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_rotation,Ncells*sizeof(float)) ); //allocate device memory
  
  //copy grid data into the host buffer
  for( int c=0; c<Ncells; c++ ){
    center[c] = vec3tofloat3(getCellCenter(c));
    anchor[c] = vec3tofloat3(getCellGlobalAnchor(c));
    size[c] = vec3tofloat3(getCellSize(c));
    rotation[c] = getCellRotation(c);
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_center, center, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_anchor, anchor, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_size, size, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_rotation, rotation, Ncells*sizeof(float), hipMemcpyHostToDevice) );

  free(hit_xyz);
  free(center);
  free(anchor);
  free(size);
  free(rotation);

  // Result buffer
  int* hit_vol = (int*)malloc(total_hits * sizeof(int));
  int* d_hit_vol;
  CUDA_CHECK_ERROR( hipMalloc(&d_hit_vol,total_hits*sizeof(int)) );

  dim3 dimBlock( 64, 1 );
  dim3 dimGrid( ceil(total_hits/64.f) );
  insideVolume <<< dimGrid, dimBlock >>>( total_hits, d_hit_xyz, getGridCellCount(), d_grid_size, d_grid_center, d_grid_anchor, d_grid_rotation, d_hit_vol );

  CUDA_CHECK_ERROR( hipPeekAtLastError() );
  CUDA_CHECK_ERROR( hipDeviceSynchronize() );
  
  CUDA_CHECK_ERROR( hipMemcpy(hit_vol, d_hit_vol, total_hits*sizeof(int), hipMemcpyDeviceToHost) );

  for( std::size_t s=0; s<Nscans; s++ ){
    if( hit_vol[s]>=0 ){
      std::cout << "Scan #" << s << " is inside gricell " << hit_vol[s] << std::endl;
    }
  }

  free(hit_vol);

  CUDA_CHECK_ERROR( hipFree(d_hit_vol) );
  CUDA_CHECK_ERROR( hipFree(d_hit_xyz) );
  CUDA_CHECK_ERROR( hipFree(d_grid_center) );
  CUDA_CHECK_ERROR( hipFree(d_grid_anchor) );
  CUDA_CHECK_ERROR( hipFree(d_grid_size) );
  CUDA_CHECK_ERROR( hipFree(d_grid_rotation) );  

  if( printmessages ){
    std::cout << "done." << std::endl;
  }
    
}

std::vector<helios::vec3> LiDARcloud::gapfillMisses( const int scan ){

  helios::vec3 origin = getScanOrigin(scan);
  std::vector<helios::vec3> xyz_filled;

  // Populating a hit table for each scan:
  // Column 0 - hit index; Column 1 - timestamp; Column 2 - ray zenith; Column 3 - ray azimuth
  std::vector<std::vector<double> > hit_table;
  for( size_t r=0; r<getHitCount(); r++ ){
    if( getHitScanID(r)==scan ){
      
      helios::SphericalCoord raydir = getHitRaydir(r);

      if( !doesHitDataExist(r,"timestamp") ){
	std::cerr << "ERROR (LiDARcloud::gapfillMisses): timestamp value is missing for hit " << r << ". Cannot gapfill. Skipping..." << std::endl;
	return xyz_filled;
      }
      
      double timestamp = getHitData(r,"timestamp");
      std::vector<double> data;
      data.resize(4);
      data.at(0) = float(r);
      data.at(1) = timestamp;
      data.at(2) = raydir.zenith;
      data.at(3) = raydir.azimuth;
      hit_table.push_back( data );
      
    }
  }
  
  // sorting, initial dt and dtheta calculations, and determining minimum target index in the scan

  //sort the hit table by column 1 (timestamp)
  std::sort( hit_table.begin(), hit_table.end(), LIDAR_CUDA::sortcol1 );
    
  int min_tindex = 1;
  for( size_t r=0; r<hit_table.size()-1; r++ ){
    
    //this is to figure out if target indexing uses 0 or 1 offset
    if( min_tindex==1 && doesHitDataExist(hit_table.at(r).at(0),"target_index") && doesHitDataExist(hit_table.at(r).at(0),"target_count") ){
      if( getHitData(hit_table.at(r).at(0),"target_index")==0 ){
	    min_tindex=0;
      }
    }
    
  }
  
  // getting rid of points with target index greater than the minimum
    
  int ndup_target = 0;
  // create new array without duplicate timestamps
  std::vector<std::vector<double> > hit_table_semiclean;
  for( size_t r=0; r<hit_table.size()-1; r++ ){    
    
    //only consider first hits
    if( doesHitDataExist(hit_table.at(r).at(0),"target_index") && doesHitDataExist(hit_table.at(r).at(0),"target_count") ){
      if( getHitData(hit_table.at(r).at(0),"target_index")>min_tindex ){
	ndup_target ++;
	continue;
      }
    }
    
    hit_table_semiclean.push_back(hit_table.at(r));
    
  }

  //  re-calculating dt

  std::vector<double> dt_semiclean;
  dt_semiclean.resize(hit_table_semiclean.size());
  for( size_t r=0; r<hit_table_semiclean.size()-1; r++ ){
    
    dt_semiclean.at(r) = hit_table_semiclean.at(r+1).at(1)-hit_table_semiclean.at(r).at(1);
    // set the hit index of the new array
    hit_table_semiclean.at(r).at(0) = r; 
    
  }

  //  checking for duplicate timestamps in the remaining data

  int ndup = 0;
  // create new array without duplicate timestamps
  std::vector<std::vector<double> > hit_table_clean;
  for( size_t r=0; r<hit_table_semiclean.size()-1; r++ ){    
    
    // if there are still rows with duplicate timestamps, it probably means there is no "target_index" column, but multiple hits per timestamp are still included
    // proceed using this assumption, just get rid of the rows where dt = 0 for simplicity (last hits probably are what remain). 
    if( dt_semiclean.at(r)==0){
      ndup ++;
      continue;
    }
        
    hit_table_clean.push_back(hit_table_semiclean.at(r));
  }

  // recalculate dt and dtheta with only first hits  
  std::vector<double> dt_clean;
  std::vector<float> dtheta_clean;
  dt_clean.resize(hit_table_clean.size());
  dtheta_clean.resize(hit_table_clean.size());
    
  double dt_clean_min = 1e6;
  for( size_t r=0; r<hit_table_clean.size()-1; r++ ){
    
    dt_clean.at(r) = hit_table_clean.at(r+1).at(1)-hit_table_clean.at(r).at(1);
    dtheta_clean.at(r) = hit_table_clean.at(r+1).at(2)-hit_table_clean.at(r).at(2);
    // set the hit index of the new array
    hit_table_clean.at(r).at(0) = r; 
    
    if( dt_clean.at(r)<dt_clean_min){
      dt_clean_min = dt_clean.at(r);
    }
    
  }
  
  // configuration of 2D map
  
  //reconfigure hit table into 2D (theta,phi) map
  std::vector<std::vector<std::vector<double> > > hit_table2D;
  
  int column = 0;
  hit_table2D.resize(1);
  for( size_t r=0; r<hit_table_clean.size()-1; r++ ){
    
    hit_table2D.at(column).push_back( hit_table_clean.at(r) );
    //for small scans (like the rectangle test case, this needs to change to < 0 or some smaller angle (that is larger than noise))
    // if( dtheta_clean.at(r) < 0 ){
    // for normal scans, this threshold allows for 10 degrees drops in theta within a given sweep as noise. This can be adjusted as appropriate. 
    if( dtheta_clean.at(r) < -0.1745329f ){
      column++;
      hit_table2D.resize(column+1);
    }
    
  }
  
  // calculate average dt and dtheta for subsequent points
 
  //calculate average dt
  float dt_avg = 0;
  int dt_sum = 0;
  
  //calculate the average dtheta to use for extrapolation
  float dtheta_avg = 0;
  int dtheta_sum = 0;
  
  for( int j=0; j<hit_table2D.size(); j++ ){
    for( int i=0; i<hit_table2D.at(j).size(); i++ ){
      int r = int(hit_table2D.at(j).at(i).at(0));
      if( dt_clean.at(r)>=dt_clean_min && dt_clean.at(r)<1.5*dt_clean_min ){
	dt_avg += dt_clean.at(r);
    	dt_sum ++;
    	
	//calculate the average dtheta to use for extrapolation
    	dtheta_avg += dtheta_clean.at(r);
    	dtheta_sum ++;
    	
      }
    }
  }
  
  dt_avg = dt_avg/float(dt_sum);
  //Calculate the average dtheta to use for extrapolation
  dtheta_avg = dtheta_avg/float(dtheta_sum);
  
  //identify gaps and fill
  for( int j=0; j<hit_table2D.size(); j++ ){
    for( int i=0; i<hit_table2D.at(j).size()-1; i++ ){
      
      double dt = hit_table2D.at(j).at(i+1).at(1)-hit_table2D.at(j).at(i).at(1);
      
      if( dt>1.5f*dt_clean_min ){ //missing hit(s)
	
	//calculate number of missing hits
	int Ngap = round(dt/dt_avg)-1;
	
	//fill missing points
	for( int k=1; k<=Ngap; k++ ){
	
	  float timestep = hit_table2D.at(j).at(i).at(1) + dt_avg*float(k);
	
	  //interpolate theta and phi
	  float theta = hit_table2D.at(j).at(i).at(2) + (hit_table2D.at(j).at(i+1).at(2)-hit_table2D.at(j).at(i).at(2))*float(k)/float(Ngap+1);
	  float phi = hit_table2D.at(j).at(i).at(3) + (hit_table2D.at(j).at(i+1).at(3)-hit_table2D.at(j).at(i).at(3))*float(k)/float(Ngap+1);
	  if( phi>2.f*M_PI ){
	    phi = phi-2.f*M_PI;
	  }
	  
	  //calculate the (x,y,z) position of the filled hit point
	  helios::SphericalCoord spherical(1e6,0.5*M_PI-theta,phi);
	  helios::vec3 xyz = origin+helios::sphere2cart(spherical);
	  
	  xyz_filled.push_back(xyz);
	
	}
      
      }
    }

  }

  /// /extrapolate missing points
  helios::vec2 theta_range = getScanRangeTheta(scan);
  
  for( int j=0; j<hit_table2D.size(); j++ ){

    //upward edge points
    if( hit_table2D.at(j).front().at(2)>theta_range.x ){

      //  float dtheta = hit_table2D.at(j).at(1).at(2)-hit_table2D.at(j).at(0).at(2);
      float dtheta = dtheta_avg;
      float theta = hit_table2D.at(j).at(0).at(2) - dtheta;
      // this commented out section extrapolates based on the last dphi - this can cause intersections of the different sweeps
      // float dphi = hit_table2D.at(j).at(1).at(3)-hit_table2D.at(j).at(0).at(3);
      //float phi = hit_table2D.at(j).at(0).at(3) - dphi;
      //just use the the last value of phi in the sweep instead
      float phi =  hit_table2D.at(j).at(0).at(3);

      if( dtheta==0 ){
	continue;
      }
      
      while( theta>theta_range.x ){
	
	helios::SphericalCoord spherical(50,0.5*M_PI-theta,phi);
	helios::vec3 xyz = origin+helios::sphere2cart(spherical);
	
	xyz_filled.push_back(xyz);
	
	theta = theta - dtheta;
	// only needed if extrapolating based on dphi above
	//phi = phi - dphi;

      }
    }

    //downward edge points
    //ERK changed .x to .y here
    if( hit_table2D.at(j).back().at(2)<theta_range.y ){
      
      int sz = hit_table2D.at(j).size();
      // same concept as above for downward edge points
      // float dtheta = hit_table2D.at(j).at(sz-1).at(2)-hit_table2D.at(j).at(sz-2).at(2);
      float dtheta = dtheta_avg;
      float theta = hit_table2D.at(j).at(sz-1).at(2) + dtheta;
      float dphi = hit_table2D.at(j).at(sz-1).at(3)-hit_table2D.at(j).at(sz-2).at(3);
      //float phi = hit_table2D.at(j).at(sz-1).at(3) + dphi;
      float phi = hit_table2D.at(j).at(sz-1).at(3);
      while( theta<theta_range.y ){
	
	helios::SphericalCoord spherical(50,0.5*M_PI-theta,phi);
	helios::vec3 xyz = origin+helios::sphere2cart(spherical);
	
	xyz_filled.push_back(xyz);
	
	theta = theta + dtheta;
	// same concept as above for downward edge points
	//	phi = phi + dphi;
	
      }
    }
    
  }
  
  return xyz_filled;
    
}
  

void LiDARcloud::calculateLeafAreaGPU( void ){
  calculateLeafAreaGPU( 1 );
}

void LiDARcloud::calculateLeafAreaGPU( const int minVoxelHits ){

  if( printmessages ){
    std::cout << "Calculating leaf area..." << std::endl;
  }
    
  if( !triangulationcomputed ){
    std::cerr << "ERROR (calculateLeafAreaGPU): Triangulation must be performed prior to leaf area calculation. See triangulateHitPoints()." << std::endl;
    exit(EXIT_FAILURE);
  }

  if( !hitgridcellcomputed ){
    calculateHitGridCellGPU();
  }
  
  const uint Nscans = getScanCount();
  const uint Ncells = getGridCellCount();

  //variable aggregates over all scans where we just keep tacking hits on the end for all scans
  std::vector<std::vector<float> > dr_agg; //dr is path length through grid cell
  dr_agg.resize(Ncells);
  std::vector<float> G_agg; //G is dot product between ray direction and triangle normal that was hit (only relevant for hits)
  G_agg.resize(Ncells,0);
  std::vector<float> hit_before_agg; //hit_before corresponds to scan points that hit something before encountering a particular grid cell
  hit_before_agg.resize(Ncells,0);
  std::vector<float> hit_after_agg; //hit_after corresponds to scan points that hit something after encountering a particular grid cell (including something inside that cell)
  hit_after_agg.resize(Ncells,0);
  std::vector<float> hit_inside_agg; //hit_inside corresponds to scan points that hit something within a particular grid cell.
  hit_inside_agg.resize(Ncells,0);
  
  //average G(theta)
  std::vector<float> Gtheta_bar;
  Gtheta_bar.resize(Ncells,0.f);
  
  // We are going to perform all calculations on a scan-by-scan basis: loop through each scan
  for( uint s=0; s<Nscans; s++ ){

    //----------- Calculate dr and number of rays passed through voxels -------------- //

    const int Nt = getScanSizeTheta(s);
    const int Np = getScanSizePhi(s);
    const size_t Nmisses = Nt*Np;

    const float3 origin = vec3tofloat3(getScanOrigin(s));

    // ----- BOUNDING BOX INTERSECTION ----- //
    // First, we are going to perform a preliminary ray trace to filter out rays that do not intersect any grid volumes.  This will speed up calculations overall. Consider all rays in scan to start.

    float3* scan_xyz = (float3*)malloc( Nmisses*sizeof(float3) );
    float* scan_weight = (float*)malloc( Nmisses*sizeof(float) );
    float3 xyz;

    //populate misses
    for( int j=0; j<Np; j++ ){
      for( int i=0; i<Nt; i++ ){
    	float3 direction = vec3tofloat3(sphere2cart(scans.at(s).rc2direction(i,j)));
    	xyz = origin + direction*10000.f;
  	
    	scan_xyz[j*Nt+i] = xyz;

	scan_weight[j*Nt+i] = 1.f;
      }
    }
    
    float3* d_scan_xyz;
    CUDA_CHECK_ERROR( hipMalloc((float3**)&d_scan_xyz, Nmisses*sizeof(float3)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_xyz, scan_xyz, Nmisses*sizeof(float3), hipMemcpyHostToDevice) );

    float* d_scan_weight;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_scan_weight, Nmisses*sizeof(float)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_weight, scan_weight, Nmisses*sizeof(float), hipMemcpyHostToDevice) );

    //set up an axis-aligned bounding box that encompasses all grid cells
    uint* boundingbox_hit = (uint*)malloc( Nmisses * sizeof(uint));
    uint* d_boundingbox_hit;
    CUDA_CHECK_ERROR( hipMalloc((uint**)&d_boundingbox_hit, Nmisses*sizeof(uint)) );
    CUDA_CHECK_ERROR( hipMemset( d_boundingbox_hit, 0, Nmisses*sizeof(uint)) ); //initialize to zero, set equal to 1 if the ray is found to intersect bounding box
    
    helios::vec3 gboxmin, gboxmax;
    getGridBoundingBox( gboxmin, gboxmax );

    float3 bbcenter = vec3tofloat3( gboxmin + 0.5*(gboxmax-gboxmin) );
    float3 bbsize = vec3tofloat3( gboxmax-gboxmin );

    //perform the ray-trace and mark all rays that passed through the bounding box
    uint3 dimBlock = make_uint3( 512, 1, 1 );
    uint3 dimGrid = make_uint3( ceil(float(Nmisses)/float(dimBlock.x)), 1, 1 );
    intersectBoundingBox <<< dimGrid, dimBlock >>>( Nmisses, origin, d_scan_xyz, bbcenter, bbsize, d_boundingbox_hit );

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here
    
    //copy results back to host
    CUDA_CHECK_ERROR( hipMemcpy( boundingbox_hit, d_boundingbox_hit, Nmisses*sizeof(uint), hipMemcpyDeviceToHost));

    //counting up how many points passed through bounding box
    size_t Nmissesbb = 0;
    for( size_t i=0; i<Nmisses; i++ ){
      if( boundingbox_hit[i]==1 ){
  	Nmissesbb ++;
      }
    }

    if( Nmissesbb==0 ){
      std::cerr << "ERROR (calculateLeafAreaGPU): No scan rays passed through grid cells.  Check that scan configuration is properly specified." << std::endl;
      std::cerr << "Terminating leaf area calculation." << std::endl;
      for( uint c=0; c<Ncells; c++ ){
	setCellLeafArea(0,c);
      }
      return;
    }
    
    CUDA_CHECK_ERROR( hipFree(d_boundingbox_hit) );
    CUDA_CHECK_ERROR( hipFree(d_scan_xyz) );

    // Reconfigure scan_xyz with points that do not hit the bounding box eliminated
    float3* missesbb_xyz = (float3*)malloc( Nmissesbb*sizeof(float3) );
    size_t miss_count=0;
    for( size_t i=0; i<Nmisses; i++ ){
      if( boundingbox_hit[i]==1 ){
	missesbb_xyz[miss_count] = scan_xyz[i];
	miss_count++;
      }
    }
    CUDA_CHECK_ERROR( hipMalloc((float3**)&d_scan_xyz, Nmissesbb*sizeof(float3)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_xyz, missesbb_xyz, Nmissesbb*sizeof(float3), hipMemcpyHostToDevice) );
    free( missesbb_xyz );
    free( boundingbox_hit );

    //Result buffer (path lengths through volume)
    float* dr = (float*)malloc( Nmissesbb * sizeof(float));
    float* d_dr;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_dr, Nmissesbb*sizeof(float)) );

    float* hit_before = (float*)malloc( sizeof(float));
    float* hit_after = (float*)malloc( sizeof(float));
    float* d_hit_before;
    float* d_hit_after;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_hit_before, sizeof(float)) );
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_hit_after, sizeof(float)) );
    
    // Perform ray-volume intersection tests to determine volumes that rays passed through
    for( uint c=0; c<Ncells; c++ ){
      
      //load the attributes of the grid cell
      float3 center = vec3tofloat3(getCellCenter(c));
      float3 anchor = vec3tofloat3(getCellGlobalAnchor(c));
      float3 size = vec3tofloat3(getCellSize(c));
      float rotation = getCellRotation(c);

      CUDA_CHECK_ERROR( hipMemset( d_dr, 0.f, Nmissesbb*sizeof(float)) );
      CUDA_CHECK_ERROR( hipMemset( d_hit_before, 0.f, sizeof(float)) );
      CUDA_CHECK_ERROR( hipMemset( d_hit_after, 0.f, sizeof(float)) );

      dimBlock = make_uint3( 512, 1, 1 );
      dimGrid = make_uint3( ceil(float(Nmissesbb)/dimBlock.x), 1, 1  );
      intersectGridcell <<< dimGrid, dimBlock >>>( Nmissesbb, origin, d_scan_xyz, d_scan_weight, center, anchor, size, rotation, d_dr, d_hit_before, d_hit_after );

       hipDeviceSynchronize();
       CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here

       //copy results back to host
       CUDA_CHECK_ERROR( hipMemcpy( dr, d_dr, Nmissesbb*sizeof(float), hipMemcpyDeviceToHost));
       CUDA_CHECK_ERROR( hipMemcpy( hit_after, d_hit_after, sizeof(float), hipMemcpyDeviceToHost));

       for( size_t r=0; r<Nmissesbb; r++ ){
       	 if( dr[r]>0.f ){
       	   dr_agg.at(c).push_back(dr[r]);
       	 }
       }
       hit_after_agg.at(c) += *hit_after;

    }

    // Perform ray-volume intersection to determine rays that intersected a leaf BEFORE passing through the volume

    std::vector<helios::vec3> this_scan_xyz;
    std::vector<float> this_scan_weight;

    for( size_t r=0; r<getHitCount(); r++ ){
      if( getHitScanID(r)==s ){
	this_scan_xyz.push_back( getHitXYZ(r) );

	if( doesHitDataExist(r,"target_count") ){
	  this_scan_weight.push_back( 1.f/getHitData(r,"target_count") );
	}else{
	  this_scan_weight.push_back(1.f);
	}
	
      }
    }

    size_t Nhits = this_scan_xyz.size();

    free( scan_xyz );
    scan_xyz = (float3*)malloc( Nhits*sizeof(float3) );
    free( scan_weight );
    scan_weight = (float*)malloc( Nhits*sizeof(float) );
   
    for( size_t r=0; r<Nhits; r++ ){
      scan_xyz[r] = vec3tofloat3(this_scan_xyz.at(r));
      scan_weight[r] = this_scan_weight.at(r);
    }

    CUDA_CHECK_ERROR( hipFree(d_scan_xyz) );
    CUDA_CHECK_ERROR( hipMalloc((float3**)&d_scan_xyz, Nhits*sizeof(float3)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_xyz, scan_xyz, Nhits*sizeof(float3), hipMemcpyHostToDevice) );

    CUDA_CHECK_ERROR( hipFree(d_scan_weight) );
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_scan_weight, Nhits*sizeof(float)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_weight, scan_weight, Nhits*sizeof(float), hipMemcpyHostToDevice) );

    CUDA_CHECK_ERROR( hipFree(d_dr) );
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_dr, Nhits*sizeof(float)) );

    for( uint c=0; c<Ncells; c++ ){
      
      //load the attributes of the grid cell
      float3 center = vec3tofloat3(getCellCenter(c));
      float3 anchor = vec3tofloat3(getCellGlobalAnchor(c));
      float3 size = vec3tofloat3(getCellSize(c));
      float rotation = getCellRotation(c);

      CUDA_CHECK_ERROR( hipMemset( d_dr, 0, Nhits*sizeof(float)) );
      CUDA_CHECK_ERROR( hipMemset( d_hit_before, 0, sizeof(float)) );
      CUDA_CHECK_ERROR( hipMemset( d_hit_after, 0, sizeof(float)) );

      dimBlock = make_uint3( min(size_t(512),Nhits), 1, 1 );
      dimGrid = make_uint3( ceil(float(Nhits)/dimBlock.x), 1, 1  );
      
      if( dimBlock.x==0 && dimGrid.x==0 ){
	continue;
      }
      
      intersectGridcell <<< dimGrid, dimBlock >>>( Nhits, origin, d_scan_xyz, d_scan_weight, center, anchor, size, rotation, d_dr, d_hit_before, d_hit_after );

      hipDeviceSynchronize();
      CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here

      //copy results back to host
      CUDA_CHECK_ERROR( hipMemcpy( hit_before, d_hit_before, sizeof(float), hipMemcpyDeviceToHost));
      
      hit_before_agg.at(c) += *hit_before;

    }
    
    free( scan_xyz );
    free( scan_weight );
    free( dr );
    free( hit_before );
    free( hit_after );
    CUDA_CHECK_ERROR( hipFree(d_scan_xyz) );
    CUDA_CHECK_ERROR( hipFree(d_scan_weight) );
    CUDA_CHECK_ERROR( hipFree(d_dr) );
    CUDA_CHECK_ERROR( hipFree(d_hit_before) );
    CUDA_CHECK_ERROR( hipFree(d_hit_after) );

 
  }//end scan loop

  //----------- Calculate number of hits in voxels -------------- //

  //figure out hits for all scans
  for( size_t r=0; r<getHitCount(); r++ ){
    if( getHitGridCell(r)>=0 ){
      helios::vec3 direction = getHitXYZ(r)-getScanOrigin(getHitScanID(r));
      direction.normalize();
      hit_inside_agg.at(getHitGridCell(r)) += sin(acos_safe(direction.z));
    }
  }
  
  //---------------------- Calculate G(theta) --------------------------//

  std::vector<float> Gtheta;
  Gtheta.resize(Ncells,0.f);

  const size_t Ntri = getTriangleCount();

  std::vector<float> denom_sum;
  denom_sum.resize(Ncells,0.f);
  std::vector<uint> cell_tri_count;
  cell_tri_count.resize(Ncells,0);
  
  for( size_t t=0; t<Ntri; t++ ){
    
    Triangulation tri = getTriangle(t);
    
    int cell = tri.gridcell;
    
    if( cell>=0 && cell<Ncells ){ //triangle is inside a grid cell
      
      helios::vec3 t0 = tri.vertex0;
      helios::vec3 t1 = tri.vertex1;
      helios::vec3 t2 = tri.vertex2;
      
      helios::vec3 v0 = t1-t0;
      helios::vec3 v1 = t2-t0;
      helios::vec3 v2 = t2-t1;
      
      float L0 = v0.magnitude();
      float L1 = v1.magnitude();
      float L2 = v2.magnitude();
      
      float S = 0.5f*(L0+L1+L2);
      float area = sqrt( S*(S-L0)*(S-L1)*(S-L2) );
      
      helios::vec3 normal = cross( v0, v2 );
      normal.normalize();
      
      helios::vec3 raydir = t0-getScanOrigin( tri.scanID );
      raydir.normalize();

      float theta = fabs(acos_safe(raydir.z));

      if( area==area ){ //in rare cases you can get area=NaN
	
	Gtheta.at(cell) += fabs(normal*raydir)*area*fabs(sin(theta));
      
	denom_sum.at(cell) += fabs(sin(theta))*area;
	cell_tri_count.at(cell) += 1;

      }
      
    }	
  }
  for( uint v=0; v<Ncells; v++ ){
    if( cell_tri_count[v]>0 ){
      Gtheta[v] = Gtheta[v]/denom_sum[v];
      Gtheta_bar[v] += Gtheta[v]/float(Nscans);
    }
  }
  

  //------------------ Perform inversion to get LAD --------------------//

  if( printmessages ){
    std::cout << "Inverting to find LAD..." << std::flush;
  }
    
  float etol = 5e-5;
  uint maxiter = 100;
    
  float error, eold, aold, tmp;
  for( uint v=0; v<Ncells; v++ ){
      
    if( Gtheta[v]==0 || Gtheta[v]!=Gtheta[v] ){
      // if( printmessages ){
      // 	std::cout << "G(theta) value bad for cell " << v << ": " << Gtheta[v] << std::endl;
      // }
      setCellLeafArea(0,v);
      continue;
    }else if( hit_after_agg[v]-hit_before_agg[v]<0 ){
      if( printmessages ){
	std::cout << "Negative number of rays reaching cell " << v << ": " << hit_after_agg[v] << " " << hit_before_agg[v] << std::endl;
      }
      setCellLeafArea(0,v);
      continue;
    }else if( hit_inside_agg[v]<minVoxelHits ){
      if( printmessages ){
	std::cout << "Not enough hits in voxel: " << hit_inside_agg[v] << " < " << minVoxelHits << std::endl;
      }
      setCellLeafArea(0,v);
      continue;
    }
      
    float P = 1.f-float(hit_inside_agg[v])/float(hit_after_agg[v]-hit_before_agg[v]);

     //assert(P>0);
      
    //initial guesses
    float a = 0.1f;
    float h = 0.01f;
    
    float mean = 0.f;
    uint count=0;
    for( int j=0; j<dr_agg.at(v).size(); j++ ){
      mean += exp(-a*dr_agg.at(v).at(j)*Gtheta[v]);
      count++;
      }
    mean /= float(dr_agg.at(v).size());      
    error = fabs(mean-P)/P;
    
    tmp = a;
    
    a = a + h;
    
    uint iter = 0;
    while( error>etol && iter<maxiter){
      
      aold = tmp;
      eold = error;
      
      float mean = 0.f;
      uint count = 0;
      for( int j=0; j<dr_agg.at(v).size(); j++ ){
	mean += exp(-a*dr_agg.at(v).at(j)*Gtheta[v]);
	count ++;
      }
      assert( count!=0 );
      mean /= float(count);      
      error = fabs(mean-P)/P;
      
      tmp = a;
      
      if( error==eold ){
	break;
      }
      
      a = fabs( (aold*error-a*eold)/(error-eold) );
      
      iter++;
      
    }
    
    float dr_bar = 0;
    for( uint i=0; i<dr_agg.at(v).size(); i++ ){
      dr_bar += dr_agg.at(v).at(i);
    }
    dr_bar /= float(dr_agg.at(v).size());
    
    if( iter>=maxiter-1 || a!=a || a>100 ){
      if( printmessages ){
	std::cout << "WARNING: LAD inversion failed for volume #" << v << ". Using average dr formulation." << std::endl;
      }
      
      a = (1.f-P)/(dr_bar*Gtheta[v]);
      
    }

    if( a>5 ){
      a = fmin((1.f-P)/dr_bar/Gtheta[v],-log(P)/dr_bar/Gtheta[v]);
    }

    helios::vec3 gridsize = getCellSize(v);
    setCellLeafArea(a*gridsize.x*gridsize.y*gridsize.z,v);

    setCellGtheta( Gtheta[v], v );

    if( printmessages ){
      std::cout << "Vol #" << v << " mean dr: " << dr_bar << std::endl;
      std::cout << "Vol #" << v << " mean G(theta): " << Gtheta[v] << std::endl;
      //std::cout << "Vol #" << v << " intersections: " << hit_inside_agg[v] << " " << hit_after_agg[v] << " " << hit_before_agg[v] << std::endl;
      
      std::cout << "Vol #" << v << " LAD: " << a << std::endl;
      //std::cout << "Vol #" << v << " LAD: " << a << std::endl;
      //   cout << "Vol #" << v << " LAD: " << (1.f-P)/(dr_bar[v]*Gthetav]) << " [point quadrat]" << endl;
      //std::cout << "Vol #" << v << " Leaf Area: " << getCellLeafArea(v)*100.f*100.f << " [P = mean(exp(-a*dr*Gtheta))]" << std::endl;
      //std::cout << "Vol #" << v << " Leaf Area: " << -log(P)/(dr_bar*Gtheta[v])*gridsize.x*gridsize.y*gridsize.z*10000.f << " [a = -ln(P)/(dr*Gtheta)]" << std::endl;
      // A_sum += LAD[v]*gridsize.x*gridsize.y*gridsize.z*100.f*100.f;
    }
      
  }

  if( printmessages ){
    std::cout << "done." << std::endl;
  }
    
}

void LiDARcloud::calculateLeafAreaGPU_testing( const int minVoxelHits ){

  if( printmessages ){
    std::cout << "Calculating leaf area..." << std::endl;
  }
    
  if( !triangulationcomputed ){
    std::cerr << "ERROR (calculateLeafAreaGPU): Triangulation must be performed prior to leaf area calculation. See triangulateHitPoints()." << std::endl;
    exit(EXIT_FAILURE);
  }

  if( !hitgridcellcomputed ){
    calculateHitGridCellGPU();
  }
  
  const uint Nscans = getScanCount();
  const uint Ncells = getGridCellCount();

  //variable aggregates over all scans where we just keep tacking hits on the end for all scans
  std::vector<std::vector<float> > dr_agg; //dr is path length through grid cell
  dr_agg.resize(Ncells);
  std::vector<float> G_agg; //G is dot product between ray direction and triangle normal that was hit (only relevant for hits)
  G_agg.resize(Ncells,0);
  std::vector<float> hit_before_agg; //hit_before corresponds to scan points that hit something before encountering a particular grid cell
  hit_before_agg.resize(Ncells,0);
  std::vector<float> hit_after_agg; //hit_after corresponds to scan points that hit something after encountering a particular grid cell (including something inside that cell)
  hit_after_agg.resize(Ncells,0);
  std::vector<float> hit_inside_agg; //hit_inside corresponds to scan points that hit something within a particular grid cell.
  hit_inside_agg.resize(Ncells,0);
  
  //average G(theta)
  std::vector<float> Gtheta_bar;
  Gtheta_bar.resize(Ncells,0.f);
  
  // We are going to perform all calculations on a scan-by-scan basis: loop through each scan
  for( uint s=0; s<Nscans; s++ ){

    // Perform ray-volume intersection to determine rays that intersected a leaf BEFORE passing through the volume

    std::vector<helios::vec3> this_scan_xyz;

    for( size_t r=0; r<getHitCount(); r++ ){
      if( getHitScanID(r)==s ){
	this_scan_xyz.push_back( getHitXYZ(r) );
      }
    }

    std::vector<helios::vec3> xyz_filled = gapfillMisses(s);
    this_scan_xyz.insert( this_scan_xyz.end(), xyz_filled.begin(), xyz_filled.end() );

    size_t Nhits = this_scan_xyz.size();

    const float3 origin = vec3tofloat3(getScanOrigin(s));

    float3* scan_xyz = (float3*)malloc( Nhits*sizeof(float3) );

    float* scan_weight = (float*)malloc( Nhits*sizeof(float) );
   
    for( size_t r=0; r<Nhits; r++ ){
      scan_xyz[r] = vec3tofloat3(this_scan_xyz.at(r));
      scan_weight[r] = 1.f;
    }

    float* hit_before = (float*)malloc( sizeof(float));
    float* hit_after = (float*)malloc( sizeof(float));
    float* d_hit_before;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_hit_before, sizeof(float)) );
    float* d_hit_after;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_hit_after, sizeof(float)) );

    float3* d_scan_xyz;
    CUDA_CHECK_ERROR( hipMalloc((float3**)&d_scan_xyz, Nhits*sizeof(float3)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_xyz, scan_xyz, Nhits*sizeof(float3), hipMemcpyHostToDevice) );

    float* d_scan_weight;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_scan_weight, Nhits*sizeof(float)) );
    CUDA_CHECK_ERROR( hipMemcpy(d_scan_weight, scan_weight, Nhits*sizeof(float), hipMemcpyHostToDevice) );

    float* dr = (float*)malloc( Nhits*sizeof(float));
    float* d_dr;
    CUDA_CHECK_ERROR( hipMalloc((float**)&d_dr, Nhits*sizeof(float)) );

    for( uint c=0; c<Ncells; c++ ){
      
      //load the attributes of the grid cell
      float3 center = vec3tofloat3(getCellCenter(c));
      float3 anchor = vec3tofloat3(getCellGlobalAnchor(c));
      float3 size = vec3tofloat3(getCellSize(c));
      float rotation = getCellRotation(c);

      CUDA_CHECK_ERROR( hipMemset( d_dr, 0, Nhits*sizeof(float)) );
      CUDA_CHECK_ERROR( hipMemset( d_hit_before, 0, sizeof(float)) );
      CUDA_CHECK_ERROR( hipMemset( d_hit_after, 0, sizeof(float)) );

      uint3 dimBlock = make_uint3( min(size_t(512),Nhits), 1, 1 );
      uint3 dimGrid = make_uint3( ceil(float(Nhits)/dimBlock.x), 1, 1  );
      
      if( dimBlock.x==0 && dimGrid.x==0 ){
	continue;
      }
      
      intersectGridcell <<< dimGrid, dimBlock >>>( Nhits, origin, d_scan_xyz, d_scan_weight, center, anchor, size, rotation, d_dr, d_hit_before, d_hit_after );

      hipDeviceSynchronize();
      CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here

      //copy results back to host
      CUDA_CHECK_ERROR( hipMemcpy( hit_before, d_hit_before, sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_ERROR( hipMemcpy( hit_after, d_hit_after, sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_ERROR( hipMemcpy( dr, d_dr, Nhits*sizeof(float), hipMemcpyDeviceToHost));
      
      hit_before_agg.at(c) += *hit_before;
      hit_after_agg.at(c) += *hit_after;

      for( size_t r=0; r<Nhits; r++ ){
	if( dr[r]>0.f ){
	  dr_agg.at(c).push_back(dr[r]);
	}
      }

    }
    
    free( scan_xyz );
    free( scan_weight );
    free( dr );
    free( hit_before );
    free( hit_after );
    CUDA_CHECK_ERROR( hipFree(d_scan_xyz) );
    CUDA_CHECK_ERROR( hipFree(d_scan_weight) );
    CUDA_CHECK_ERROR( hipFree(d_dr) );
    CUDA_CHECK_ERROR( hipFree(d_hit_before) );
    CUDA_CHECK_ERROR( hipFree(d_hit_after) );

 
  }//end scan loop

  //----------- Calculate number of hits in voxels -------------- //

  //figure out hits for all scans
  for( size_t r=0; r<getHitCount(); r++ ){
    if( getHitGridCell(r)>=0 ){
      helios::vec3 direction = getHitXYZ(r)-getScanOrigin(getHitScanID(r));
      direction.normalize();
      hit_inside_agg.at(getHitGridCell(r)) += sin(acos_safe(direction.z));
    }
  }
  
  //---------------------- Calculate G(theta) --------------------------//

  std::vector<float> Gtheta;
  Gtheta.resize(Ncells,0.f);

  const size_t Ntri = getTriangleCount();

  std::vector<float> area_sum;
  area_sum.resize(Ncells,0.f);
  std::vector<float> sin_sum;
  sin_sum.resize(Ncells,0.f);
  std::vector<uint> cell_tri_count;
  cell_tri_count.resize(Ncells,0);
  
  for( size_t t=0; t<Ntri; t++ ){
    
    Triangulation tri = getTriangle(t);
    
    int cell = tri.gridcell;
    
    if( cell>=0 && cell<Ncells ){ //triangle is inside a grid cell
      
      helios::vec3 t0 = tri.vertex0;
      helios::vec3 t1 = tri.vertex1;
      helios::vec3 t2 = tri.vertex2;
      
      helios::vec3 v0 = t1-t0;
      helios::vec3 v1 = t2-t0;
      helios::vec3 v2 = t2-t1;
      
      float L0 = v0.magnitude();
      float L1 = v1.magnitude();
      float L2 = v2.magnitude();
      
      float S = 0.5f*(L0+L1+L2);
      float area = sqrt( S*(S-L0)*(S-L1)*(S-L2) );
      
      helios::vec3 normal = cross( v0, v2 );
      normal.normalize();
      
      helios::vec3 raydir = t0-getScanOrigin( tri.scanID );
      raydir.normalize();

      float theta = fabs(acos_safe(raydir.z));

      if( area==area ){ //in rare cases you can get area=NaN
	
	Gtheta.at(cell) += fabs(normal*raydir)*area*fabs(sin(theta));
      
	area_sum.at(cell) += area;
	sin_sum.at(cell) += fabs(sin(theta));
	cell_tri_count.at(cell) += 1;

      }
      
    }	
  }
  for( uint v=0; v<Ncells; v++ ){
    if( cell_tri_count[v]>0 ){
      Gtheta[v] *= float(cell_tri_count[v])/(area_sum[v]*sin_sum[v]);
      Gtheta_bar[v] += Gtheta[v]/float(Nscans);
    }
  }
  

  //------------------ Perform inversion to get LAD --------------------//

  if( printmessages ){
    std::cout << "Inverting to find LAD..." << std::flush;
  }
    
  float etol = 5e-5;
  uint maxiter = 100;
    
  float error, eold, aold, tmp;
  for( uint v=0; v<Ncells; v++ ){
      
    if( Gtheta[v]==0 || Gtheta[v]!=Gtheta[v] ){
      // if( printmessages ){
      // 	std::cout << "G(theta) value bad for cell " << v << ": " << Gtheta[v] << std::endl;
      // }
      setCellLeafArea(0,v);
      continue;
    }else if( hit_inside_agg[v]<minVoxelHits ){
      if( printmessages ){
	std::cout << "Not enough hits in voxel: " << hit_inside_agg[v] << " < " << minVoxelHits << std::endl;
      }
      setCellLeafArea(0,v);
      continue;
    }
      
    float P = 1.f-float(hit_inside_agg[v])/float(hit_after_agg[v]);

     //assert(P>0);
      
    //initial guesses
    float a = 0.1f;
    float h = 0.01f;
    
    float mean = 0.f;
    uint count=0;
    for( int j=0; j<dr_agg.at(v).size(); j++ ){
      mean += exp(-a*dr_agg.at(v).at(j)*Gtheta[v]);
      count++;
      }
    mean /= float(dr_agg.at(v).size());      
    error = fabs(mean-P)/P;
    
    tmp = a;
    
    a = a + h;
    
    uint iter = 0;
    while( error>etol && iter<maxiter){
      
      aold = tmp;
      eold = error;
      
      float mean = 0.f;
      uint count = 0;
      for( int j=0; j<dr_agg.at(v).size(); j++ ){
	mean += exp(-a*dr_agg.at(v).at(j)*Gtheta[v]);
	count ++;
      }
      assert( count!=0 );
      mean /= float(count);      
      error = fabs(mean-P)/P;
      
      tmp = a;
      
      if( error==eold ){
	break;
      }
      
      a = fabs( (aold*error-a*eold)/(error-eold) );
      
      iter++;
      
    }
    
    float dr_bar = 0;
    for( uint i=0; i<dr_agg.at(v).size(); i++ ){
      dr_bar += dr_agg.at(v).at(i);
    }
    dr_bar /= float(dr_agg.at(v).size());
    
    if( iter>=maxiter-1 || a!=a || a>100 ){
      if( printmessages ){
	std::cout << "WARNING: LAD inversion failed for volume #" << v << ". Using average dr formulation." << std::endl;
      }
      
      a = (1.f-P)/(dr_bar*Gtheta[v]);
      
    }

    if( a>5 ){
      a = fmin((1.f-P)/dr_bar/Gtheta[v],-log(P)/dr_bar/Gtheta[v]);
    }

    helios::vec3 gridsize = getCellSize(v);
    setCellLeafArea(a*gridsize.x*gridsize.y*gridsize.z,v);

    setCellGtheta( Gtheta[v], v );

    if( printmessages ){
      std::cout << "Vol #" << v << " mean dr: " << dr_bar << std::endl;
      std::cout << "Vol #" << v << " mean G(theta): " << Gtheta[v] << std::endl;
      //std::cout << "Vol #" << v << " intersections: " << hit_inside_agg[v] << " " << hit_after_agg[v] << " " << hit_before_agg[v] << std::endl;
      
      std::cout << "Vol #" << v << " LAD: " << a << std::endl;
      //std::cout << "Vol #" << v << " LAD: " << a << std::endl;
      //   cout << "Vol #" << v << " LAD: " << (1.f-P)/(dr_bar[v]*Gthetav]) << " [point quadrat]" << endl;
      //std::cout << "Vol #" << v << " Leaf Area: " << getCellLeafArea(v)*100.f*100.f << " [P = mean(exp(-a*dr*Gtheta))]" << std::endl;
      //std::cout << "Vol #" << v << " Leaf Area: " << -log(P)/(dr_bar*Gtheta[v])*gridsize.x*gridsize.y*gridsize.z*10000.f << " [a = -ln(P)/(dr*Gtheta)]" << std::endl;
      // A_sum += LAD[v]*gridsize.x*gridsize.y*gridsize.z*100.f*100.f;
    }
      
  }

  if( printmessages ){
    std::cout << "done." << std::endl;
  }
    
}

void LiDARcloud::calculateLeafAreaGPU_synthetic( helios::Context* context, bool beamoutput, bool fillAnalytic  ){
    
    // calculates LAD using several different methods investigated in Kent & Bailey (2021)
    // writes all voxel level variables to a file
    // optionally writes detailed information about each beam
    // unlike previous calculateLeafAreaGPU versions, this one does not set the cell leaf area variable and does not filter based on minVoxelHits argument 
    
    if( printmessages ){
        std::cout << "Calculating leaf area..." << std::endl;
    }
    
    if( !triangulationcomputed ){
        std::cerr << "ERROR (calculateLeafAreaGPU): Triangulation must be performed prior to leaf area calculation. See triangulateHitPoints()." << std::endl;
        exit(EXIT_FAILURE);
    }
    
    if( !hitgridcellcomputed ){
        calculateHitGridCellGPU();
    }
    
    const uint Nscans = getScanCount();
    const uint Ncells = getGridCellCount();
    
    //variable aggregates over all scans where we just keep tacking hits on the end for all scans
    std::vector<std::vector<float> > dr_agg; //dr is path length through grid cell
    dr_agg.resize(Ncells);
    std::vector<float> G_agg; //G is dot product between ray direction and triangle normal that was hit (only relevant for hits)
    G_agg.resize(Ncells,0);
    std::vector<float> hit_before_agg; //hit_before corresponds to scan points that hit something before encountering a particular grid cell
    hit_before_agg.resize(Ncells,0);
    std::vector<float> hit_after_agg; //hit_after corresponds to scan points that hit something after encountering a particular grid cell (including something inside that cell)
    hit_after_agg.resize(Ncells,0);
    std::vector<float> hit_inside_agg; //hit_inside corresponds to scan points that hit something within a particular grid cell.
    hit_inside_agg.resize(Ncells,0);
    
    //average G(theta)
    std::vector<float> Gtheta_bar;
    Gtheta_bar.resize(Ncells,0.f);
    
    // nested vectors to save P calculation terms
    // cell, scan, variable, value
    std::vector<std::vector<float>>  P_first_numerator_array(Ncells);
    std::vector<std::vector<float>>  P_first_denominator_array(Ncells);
    
    std::vector<std::vector<float>>  P_sequal_numerator_array(Ncells);
    std::vector<std::vector<float>>  P_sequal_denominator_array(Ncells);
    
    std::vector<std::vector<float>>  P_equal_numerator_array(Ncells);
    std::vector<std::vector<float>>  P_equal_denominator_array(Ncells);
    
    std::vector<std::vector<float>>  P_intensity_numerator_array(Ncells);
    std::vector<std::vector<float>>  P_intensity_denominator_array(Ncells);
    
    std::vector<std::vector<float>>  P_exact_numerator_array(Ncells);
    std::vector<std::vector<float>>  P_exact_denominator_array(Ncells);
    
    std::vector<std::vector<uint>>  voxel_beam_count_array(Ncells);
    
    
    std::vector<std::vector<float>>  dr_array(Ncells);
    
    
    // r is the index of all hitpoints (all scans)
    // it is saved for the current scan in this_scan_index
    // i is the index of this_scan_index
    // so when accessing the things using getHitData(), getHitXYZ(), etc, 
    // for only one of the scans (this_scan), 
    // need to use getHitData(this_scan_index[i]) not getHitData(i)
    
    ////////////// We are going to perform all calculations on a scan-by-scan basis: loop through each scan
    for( uint s=0; s<Nscans; s++ ){
        
        if( printmessages ){
            std::cout << "------------------------SCAN = " << s << std::endl;
        }
        
        //only work with hitpoints assoicated with current scan
        std::vector<helios::vec3> this_scan_xyz;
        std::vector<uint> this_scan_index;
        for( size_t r=0; r<getHitCount(); r++ ){
            if( getHitScanID(r)==s ){
                this_scan_xyz.push_back( getHitXYZ(r) );
                this_scan_index.push_back(r);
            }
        }
        // size of the array of all the hits in this scan
        size_t Nhits = this_scan_xyz.size();
        
        if( printmessages ){
            std::cout << "Number of hitpoints associated with this scan = Nhits = " << Nhits << std::endl;
        }
        
        // count the number of beams (i.e. don't count multiple hits per beam)
        float previous_time = -1.f;
        uint Nbeams = 0;
        for(uint i=0; i < Nhits; i++)
        {
            float current_time = getHitData(this_scan_index[i], "timestamp");
            if(current_time != previous_time){
                Nbeams ++;
                previous_time = current_time;
            }
        }
        
        if( printmessages ){
            std::cout << "Number of beams associated with this scan = Nbeams = " << Nbeams << std::endl;
        }
        
        // create a vector each element of which represents a unique beam and is another vector of the hit point indices of this_scan_xyz that is associated with this beam
        float previous_beam = 0;
        uint beam_ID = 0;
        std::vector<std::vector<uint>> beam_array(Nbeams);
        for(uint i=0; i < Nhits; i++)
        {
            float current_beam = getHitData(this_scan_index[i], "timestamp");
            
            if(current_beam == previous_beam)
            {
                beam_array.at(beam_ID).push_back(i); //ERK 
            }else{
                beam_ID ++;
                beam_array.at(beam_ID).push_back(i); //ERK
                previous_beam = current_beam;
            }
            
        }
        
        // set up GPU things for current scan
        const float3 origin = vec3tofloat3(getScanOrigin(s));
        float3* scan_xyz = (float3*)malloc( Nhits*sizeof(float3) );
        float* scan_weight = (float*)malloc( Nhits*sizeof(float) );
        
        for( size_t i=0; i<Nhits; i++ ){
            scan_xyz[i] = vec3tofloat3(this_scan_xyz.at(i));
            scan_weight[i] = 1.f;
        }
        
        float* hit_before = (float*)malloc( sizeof(float));
        float* hit_after = (float*)malloc( sizeof(float));
        float* d_hit_before;
        CUDA_CHECK_ERROR( hipMalloc((float**)&d_hit_before, sizeof(float)) );
        float* d_hit_after;
        CUDA_CHECK_ERROR( hipMalloc((float**)&d_hit_after, sizeof(float)) );
        
        float3* d_scan_xyz;
        CUDA_CHECK_ERROR( hipMalloc((float3**)&d_scan_xyz, Nhits*sizeof(float3)) );
        CUDA_CHECK_ERROR( hipMemcpy(d_scan_xyz, scan_xyz, Nhits*sizeof(float3), hipMemcpyHostToDevice) );
        
        float* d_scan_weight;
        CUDA_CHECK_ERROR( hipMalloc((float**)&d_scan_weight, Nhits*sizeof(float)) );
        CUDA_CHECK_ERROR( hipMemcpy(d_scan_weight, scan_weight, Nhits*sizeof(float), hipMemcpyHostToDevice) );
        
        float* dr = (float*)malloc( Nhits*sizeof(float));
        float* d_dr;
        CUDA_CHECK_ERROR( hipMalloc((float**)&d_dr, Nhits*sizeof(float)) );
        
        uint* hit_location = (uint*)malloc( Nhits*sizeof(uint));
        uint* d_hit_location;
        CUDA_CHECK_ERROR( hipMalloc((uint**)&d_hit_location, Nhits*sizeof(uint)) );
        
        
        /////////////////////////////////////////////////////////////////////
        // now loop through each voxel
        for( uint c=0; c<Ncells; c++ ){
            
            if( printmessages ){
                std::cout << "----CELL = " << c << std::endl;
            }
            
            std::ofstream file_beam;
            if( beamoutput){
                // set up header of file that outputs one row for each beam in the current scan that interacts with the current voxel
                
                file_beam.open("../beamoutput/beam_data_s_" + std::to_string(s) + "_c_" + std::to_string(c) + ".txt");
                file_beam << "scan, cell, beam, R_before, R_inside, R_after, R_miss, E_before, E_inside, E_after, E_miss, sin_theta, dr, last_dr"  << std::endl;
            }
            
            //load the attributes of the grid cell
            float3 center = vec3tofloat3(getCellCenter(c));
            float3 anchor = vec3tofloat3(getCellGlobalAnchor(c));
            float3 size = vec3tofloat3(getCellSize(c));
            float rotation = getCellRotation(c);
            
            CUDA_CHECK_ERROR( hipMemset( d_hit_location, 0, Nhits*sizeof(uint)) );
            CUDA_CHECK_ERROR( hipMemset( d_dr, 0, Nhits*sizeof(float)) );
            CUDA_CHECK_ERROR( hipMemset( d_hit_before, 0, sizeof(float)) );
            CUDA_CHECK_ERROR( hipMemset( d_hit_after, 0, sizeof(float)) );
            
            uint3 dimBlock = make_uint3( min(size_t(512),Nhits), 1, 1 );
            uint3 dimGrid = make_uint3( ceil(float(Nhits)/dimBlock.x), 1, 1  );
            
            if( dimBlock.x==0 && dimGrid.x==0 ){
                continue;
            }
            
            float scanner_range = 5000.0;
            intersectGridcell_synthetic <<< dimGrid, dimBlock >>>( Nhits, origin, d_scan_xyz, d_scan_weight, center, anchor, size, rotation, d_dr, d_hit_before, d_hit_after, d_hit_location, scanner_range );
            
            hipDeviceSynchronize();
            CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here
            
            //copy results back to host
            CUDA_CHECK_ERROR( hipMemcpy( hit_before, d_hit_before, sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK_ERROR( hipMemcpy( hit_after, d_hit_after, sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK_ERROR( hipMemcpy( dr, d_dr, Nhits*sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK_ERROR( hipMemcpy( hit_location, d_hit_location, Nhits*sizeof(uint), hipMemcpyDeviceToHost));
            
            float P_first_numerator = 0;
            float P_first_denominator = 0;
            float P_sequal_numerator = 0;
            float P_sequal_denominator = 0;
            float P_equal_numerator = 0;
            float P_equal_denominator = 0;
            float P_intensity_numerator = 0;
            float P_intensity_denominator = 0;
            float P_exact_numerator = 0;
            float P_exact_denominator = 0;
            uint voxel_beam_count = 0;
            
            if( printmessages ){
                std::cout << "Nbeams = " << Nbeams << std::endl;
            }
            ////////////////////////// loop through each beam associated with the current scan
            for(int k = 0; k < Nbeams; k++){
                
                float R_before = 0;
                float R_inside = 0;
                float R_after = 0;
                float R_miss = 0;
                
                float E_before = 0;
                float E_inside = 0;
                float E_after = 0;
                float E_miss = 0;
                
                float sin_theta;
                float W = 0;
                float drr = 0;
                float last_drr = 0;
                
                ////////// loop through the hitpoints in the current beam to get number of rays in each location
                for(int j = 0; j < beam_array.at(k).size(); j++){
                    
                    // pull out the index of the current scan's current beam's current hit (which is used to access the overall hit index r through this_scan_index[i])
                    uint i = beam_array.at(k).at(j);
                    
                    helios::vec3 direction = getHitXYZ(this_scan_index[i])-getScanOrigin(getHitScanID(this_scan_index[i]));
                    direction.normalize();
                    sin_theta = sin(acos_safe(direction.z));
                    
                    last_drr = dr[i];
                    drr += dr[i];
                    
                    if(hit_location[i] == 1){
                        R_before = R_before +  getHitData(this_scan_index[i], "nRaysHit");
                        E_before ++;
                    }else if(hit_location[i] == 2){
                        R_inside = R_inside +  getHitData(this_scan_index[i], "nRaysHit");
                        E_inside ++;
                    }else if(hit_location[i] == 3){
                        R_after = R_after +  getHitData(this_scan_index[i], "nRaysHit");
                        E_after ++;
                        if(getHitData(this_scan_index[i], "target_index") == 0){ // if this is the first hitpoint for this beam,
                            W = 1.0;
                        }
                    }else if(hit_location[i] == 4){
                        R_miss = R_miss +  getHitData(this_scan_index[i], "nRaysHit");
                        E_miss ++;
                    } // or this hitpoint / beam did not intersect the voxel and should not be added to the total number of beams for this voxel
                    
                } // end of loop through each hit in the current beam
                
                // calculate the average dr across all hitpoints for the beam (they should all be the same anyway... but just in case)
                float drrx = drr / float(beam_array.at(k).size());
                
                // if the path length is greater than 0 (it passes throught the current voxel), save it as an element of
                if( drrx > 0.f )
                {
                    dr_array.at(c).push_back(drrx);
                }
                
                if( beamoutput ){
                    // output info about the current beam to file
                    file_beam << s << "," << c << "," << k << "," << R_before << "," << R_inside << "," << R_after << "," << R_miss << "," << E_before << "," << E_inside << "," << E_after << "," << E_miss << "," << sin_theta << "," << drrx << "," << last_drr << std::endl;
                }
                
                ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                // calculate terms for different P methods
                
                // P_exact
                if(R_inside != 0 || R_after != 0 || R_miss != 0){ // only count this beam for P_exact if some energy made it to the voxel
                    P_exact_numerator += ((R_after + R_miss) / (R_before + R_inside + R_after + R_miss))*sin_theta;
                    P_exact_denominator += ((R_inside + R_after + R_miss) / (R_before + R_inside + R_after + R_miss))*sin_theta;
                    voxel_beam_count++;
                }
                
                // P_intensity
                if(R_inside != 0 || R_after != 0){ // only count this beam for P_intensity if some energy hit inside or after the voxel (not including misses)
                    P_intensity_numerator += (R_after / (R_before + R_inside + R_after))*sin_theta;
                    P_intensity_denominator += ((R_inside + R_after) / (R_before + R_inside + R_after))*sin_theta;
                }else if(R_inside == 0 && R_after == 0 && R_before == 0 && R_miss != 0){ // also count this beam if all the energy missed (but still went throught the voxel)
                    P_intensity_numerator += 1*sin_theta;
                    P_intensity_denominator += 1*sin_theta;
                }
                
                // P_equal
                if(E_inside != 0 || E_after != 0){ // only count for P_equal if some hit points were inside or after the voxel
                    P_equal_numerator += (E_after / (E_before + E_inside + E_after))*sin_theta;
                    P_equal_denominator += ((E_inside + E_after) / (E_before + E_inside + E_after))*sin_theta;
                }else if(E_inside == 0 && E_after == 0 && E_before == 0 && E_miss != 0){ // also count this beam if there is only a "hitpoint" that missed (far after the voxel)
                    P_equal_numerator += 1*sin_theta;
                    P_equal_denominator += 1*sin_theta;
                }
                
                // P_sequal
                if(E_inside != 0 || E_after != 0){ // only count for P_sequal if some hit points were inside or after the voxel
                    P_sequal_numerator += E_after / (E_inside + E_after);
                    P_sequal_denominator += 1;
                }else if(E_inside == 0 && E_after == 0 && E_before == 0 && E_miss != 0){ // also count this beam if there is only a "hitpoint" that missed (far after the voxel)
                    P_sequal_numerator += 1;
                    P_sequal_denominator += 1;
                }
                
                // P_first
                if(E_before == 0 && (E_inside != 0 || E_after != 0)){ // only count for P_sequal if some hit points were inside or after the voxel
                    P_first_numerator += W*sin_theta;
                    P_first_denominator += 1*sin_theta;
                }else if(E_inside == 0 && E_after == 0 && E_before == 0 && E_miss != 0){ // also count this beam if there is only a "hitpoint" that missed (far after the voxel)
                    P_first_numerator += 1*sin_theta;
                    P_first_denominator += 1*sin_theta;
                }
                
                
            }// end of loop through all beams for the current cell
            
            // save results for the current cell to the arrays
            
            P_first_numerator_array.at(c).push_back(P_first_numerator);
            P_first_denominator_array.at(c).push_back(P_first_denominator);
            
            P_sequal_numerator_array.at(c).push_back(P_sequal_numerator);
            P_sequal_denominator_array.at(c).push_back(P_sequal_denominator);
            
            P_equal_numerator_array.at(c).push_back(P_equal_numerator);
            P_equal_denominator_array.at(c).push_back(P_equal_denominator);
            
            P_intensity_numerator_array.at(c).push_back(P_intensity_numerator);
            P_intensity_denominator_array.at(c).push_back(P_intensity_denominator);
            
            P_exact_numerator_array.at(c).push_back(P_exact_numerator);
            P_exact_denominator_array.at(c).push_back(P_exact_denominator);
            
            voxel_beam_count_array.at(c).push_back(voxel_beam_count);
            
            hit_before_agg.at(c) += *hit_before;
            hit_after_agg.at(c) += *hit_after;
            
            for( size_t i=0; i<Nhits; i++ ){
                if( dr[i]>0.f ){
                    dr_agg.at(c).push_back(dr[i]);
                }
            }
            
            file_beam.close();
        } // end of loop through all cells
        
        free( scan_xyz );
        free( scan_weight );
        free( dr );
        free( hit_before );
        free( hit_after );
        CUDA_CHECK_ERROR( hipFree(d_scan_xyz) );
        CUDA_CHECK_ERROR( hipFree(d_scan_weight) );
        CUDA_CHECK_ERROR( hipFree(d_dr) );
        CUDA_CHECK_ERROR( hipFree(d_hit_before) );
        CUDA_CHECK_ERROR( hipFree(d_hit_after) );
        
    }//end scan loop
    
    std::vector<float> P_first(Ncells);
    std::vector<float> P_sequal(Ncells);
    std::vector<float> P_equal(Ncells);
    std::vector<float> P_intensity(Ncells);
    std::vector<float> P_exact(Ncells);
    std::vector<uint> voxel_beam_count_tot(Ncells);
    std::vector<float> dr_bar(Ncells);
    
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // now aggregate over all scans
    for( uint c=0; c<Ncells; c++ ){
        
        float P_first_numerator_array_agg = 0;
        float P_first_denominator_array_agg = 0;
        
        float P_sequal_numerator_array_agg = 0;
        float P_sequal_denominator_array_agg = 0;
        
        float P_equal_numerator_array_agg = 0;
        float P_equal_denominator_array_agg = 0;
        
        float P_intensity_numerator_array_agg= 0;
        float P_intensity_denominator_array_agg= 0;
        
        float P_exact_numerator_array_agg= 0;
        float P_exact_denominator_array_agg= 0;
        
        uint voxel_beam_count_array_agg=0;
        
        
        for( uint s=0; s<Nscans; s++ ){
            
            P_first_numerator_array_agg += P_first_numerator_array.at(c).at(s);
            P_first_denominator_array_agg += P_first_denominator_array.at(c).at(s);
            
            P_sequal_numerator_array_agg += P_sequal_numerator_array.at(c).at(s);
            P_sequal_denominator_array_agg += P_sequal_denominator_array.at(c).at(s);
            
            P_equal_numerator_array_agg += P_equal_numerator_array.at(c).at(s);
            P_equal_denominator_array_agg += P_equal_denominator_array.at(c).at(s);
            
            P_intensity_numerator_array_agg += P_intensity_numerator_array.at(c).at(s);
            P_intensity_denominator_array_agg += P_intensity_denominator_array.at(c).at(s);
            
            P_exact_numerator_array_agg += P_exact_numerator_array.at(c).at(s);
            P_exact_denominator_array_agg += P_exact_denominator_array.at(c).at(s);
            voxel_beam_count_array_agg += voxel_beam_count_array.at(c).at(s);
        }
        
        
        P_first[c] = P_first_numerator_array_agg / P_first_denominator_array_agg ;
        P_sequal[c] = P_sequal_numerator_array_agg / P_sequal_denominator_array_agg ;
        P_equal[c] = P_equal_numerator_array_agg / P_equal_denominator_array_agg ;
        P_intensity[c] = P_intensity_numerator_array_agg / P_intensity_denominator_array_agg ;
        P_exact[c] = P_exact_numerator_array_agg / P_exact_denominator_array_agg ;
        voxel_beam_count_tot[c] = voxel_beam_count_array_agg;
        if( printmessages ){
            std::cout << "Cell " << c << ", voxel_beam_count = " << voxel_beam_count_tot[c]  << std::endl;
            std::cout << "Cell " << c << ", P_first = " << P_first[c] << std::endl;
            std::cout << "Cell " << c << ", P_sequal = " << P_sequal[c] << std::endl;
            std::cout << "Cell " << c << ", P_equal = " << P_equal[c] << std::endl;
            std::cout << "Cell " << c << ", P_intensity = " << P_intensity[c] << std::endl;
            std::cout << "Cell " << c << ", P_exact = " << P_exact[c] << std::endl;
        }
        
        float mean_dr = 0;
        for(int j = 0; j < dr_array.at(c).size(); j++)
        {
            mean_dr += dr_array.at(c).at(j);
        }
        dr_bar[c] = mean_dr/ float(dr_array.at(c).size());
        if( printmessages ){
            std::cout << "Cell " << c << ", dr_bar = " <<  dr_bar[c] << std::endl;
        }
    } //end of loop through cells to aggregate all scans
    
    
    /// old code in original calculateLeafAreaGPU_testing() ... do we need to keep this??
    //----------- Calculate number of hits in voxels -------------- //
    
    //figure out hits for all scans
    for( size_t r=0; r<getHitCount(); r++ ){
        if( getHitGridCell(r)>=0 ){
            helios::vec3 direction = getHitXYZ(r)-getScanOrigin(getHitScanID(r));
            direction.normalize();
            hit_inside_agg.at(getHitGridCell(r)) += sin(acos_safe(direction.z));
        }
    }
    
    //---------------------- Calculate G(theta) from triangulation --------------------------//
    
    std::vector<float> Gtheta;
    Gtheta.resize(Ncells,0.f);
    
    const size_t Ntri = getTriangleCount();
    
    std::vector<uint> cell_tri_count;
    cell_tri_count.resize(Ncells,0);
    
    std::vector<float> area_sin_sum;
    area_sin_sum.resize(Ncells,0.f);
    
    for( size_t t=0; t<Ntri; t++ ){
        
        Triangulation tri = getTriangle(t);
        
        int cell = tri.gridcell;
        
        if( cell>=0 && cell<Ncells ){ //triangle is inside a grid cell
            
            helios::vec3 t0 = tri.vertex0;
            helios::vec3 t1 = tri.vertex1;
            helios::vec3 t2 = tri.vertex2;
            
            helios::vec3 v0 = t1-t0;
            helios::vec3 v1 = t2-t0;
            helios::vec3 v2 = t2-t1;
            
            float L0 = v0.magnitude();
            float L1 = v1.magnitude();
            float L2 = v2.magnitude();
            
            float S = 0.5f*(L0+L1+L2);
            float area = sqrt( S*(S-L0)*(S-L1)*(S-L2) );
            
            helios::vec3 normal = cross( v0, v2 );
            normal.normalize();
            
            helios::vec3 raydir = t0-getScanOrigin( tri.scanID );
            raydir.normalize();
            
            float theta = fabs(acos_safe(raydir.z));
            
            if( area==area ){ //in rare cases you can get area=NaN
                
                Gtheta.at(cell) += fabs(normal*raydir)*area*fabs(sin(theta));
                
                area_sin_sum.at(cell) += area*fabs(sin(theta));
                cell_tri_count.at(cell) += 1;
                
            }
            
        }
    }
    
    for( uint c=0; c<Ncells; c++ ){
        if( cell_tri_count[c]>0 ){
            //Gtheta[c] *= float(cell_tri_count[c])/(area_sum[c]*sin_sum[c]);
            //Gtheta[c] *= float(cell_tri_count[c])/(area_sin_sum[c]);
            Gtheta[c] *= 1.0/(area_sin_sum[c]);
            
            Gtheta_bar[c] += Gtheta[c]/float(Nscans);
        }
    }
    
    
    //------------------Calculate "Reference" P and also mean and variance of dr --------------------//
    // this is the value of P calculated using the reference values of LAD and Gtheta (from calculateSyntheticLeafArea, calculateSyntheticGtheta),
    // and the path length from all beams for a given cell
    
    // first get the reference LA and Gtheta using modified functions that save output to arrays
    std::vector<float> LA_ref = calculateSyntheticLeafArea(context);
    std::vector<float> Gtheta_ref = calculateSyntheticGtheta(context);
    
    std::vector<float> P_ref(Ncells);
    P_ref.resize(Ncells,0.f);
    
    std::vector<float> dr_bar_ref(Ncells);
    dr_bar_ref.resize(Ncells,0.f);
    
    std::vector<float> dr_var_ref(Ncells);
    dr_var_ref.resize(Ncells,0.f);
    
    for(uint c=0; c<Ncells; c++)
    {
        helios::vec3 gridsize = getCellSize(c);
        float LAD_ref = LA_ref.at(c) / (gridsize.x*gridsize.y*gridsize.z);
        float G_ref = Gtheta_ref.at(c);
        
        float dr_bar_tmp = 0;
        float sum_term = 0;
        for(uint j=0; j<dr_array.at(c).size(); j++)
        {
            dr_bar_tmp += dr_array.at(c).at(j);
            sum_term += exp(-1.0*LAD_ref*G_ref*dr_array.at(c).at(j));
        }
        sum_term /= float(dr_array.at(c).size());
        dr_bar_tmp /= float(dr_array.at(c).size());
        
        dr_bar_ref[c] = dr_bar_tmp;
        P_ref[c] = sum_term;
    }
    
    for(uint c=0; c<Ncells; c++)
    {
        float dr_var = 0;
        for(uint j=0; j<dr_array.at(c).size(); j++)
        {
            dr_var += pow(dr_array.at(c).at(j) - dr_bar_ref[c], 2.0);
        }
        dr_var /= float(dr_array.at(c).size());
        dr_var_ref[c] = dr_var;
    }
    
    if( printmessages ){
        std::cout << "finished P_ref and dr calculations" << std::endl;
    }
    
    //------------------ Perform inversions to get LAD using the different methods for P --------------------//
    
    if( printmessages ){
        std::cout << "Inverting to find LAD..." << std::flush;
    }
    
    // perform LAD inversion using reference Gtheta
    std::vector<float> LAD_refcheck_Gref = LAD_inversion(P_ref, Gtheta_ref, dr_array, fillAnalytic);
    std::vector<float> LAD_first_Gref = LAD_inversion(P_first, Gtheta_ref, dr_array, fillAnalytic);
    std::vector<float> LAD_sequal_Gref = LAD_inversion(P_sequal, Gtheta_ref, dr_array, fillAnalytic);
    std::vector<float> LAD_equal_Gref = LAD_inversion(P_equal, Gtheta_ref, dr_array, fillAnalytic);
    std::vector<float> LAD_intensity_Gref = LAD_inversion(P_intensity, Gtheta_ref, dr_array, fillAnalytic);
    std::vector<float> LAD_exact_Gref = LAD_inversion(P_exact, Gtheta_ref, dr_array, fillAnalytic );
    
    // perform LAD inversion using triangulation-estimated Gtheta
    std::vector<float> LAD_refcheck = LAD_inversion(P_ref, Gtheta, dr_array, fillAnalytic);
    std::vector<float> LAD_first = LAD_inversion(P_first, Gtheta, dr_array, fillAnalytic);
    std::vector<float> LAD_sequal = LAD_inversion(P_sequal, Gtheta, dr_array, fillAnalytic);
    std::vector<float> LAD_equal = LAD_inversion(P_equal, Gtheta, dr_array, fillAnalytic);
    std::vector<float> LAD_intensity = LAD_inversion(P_intensity, Gtheta, dr_array, fillAnalytic);
    std::vector<float> LAD_exact = LAD_inversion(P_exact, Gtheta, dr_array, fillAnalytic);
    
    if( printmessages ){
        std::cout << "finished LAD inversions" << std::endl;
    }
    
    // output the voxel level variables
    std::ofstream file_output;
    file_output.open("../voxeloutput/voxeloutput.txt");
    file_output << "cell, grid_center_x, grid_center_y, grid_center_z, grid_size_x, grid_size_y, grid_size_z, Nbeams, LAD_ref, G_ref, dr_bar, dr_var, P_ref, G, P_first, P_sequal, P_equal, P_intensity, P_exact, LADGref_refcheck, LADGref_first, LADGref_sequal, LADGref_equal, LADGref_intensity, LADGref_exact, LAD_refcheck, LAD_first, LAD_sequal, LAD_equal, LAD_intensity, LAD_exact"  << std::endl;
    
    for(uint c=0; c<Ncells; c++)
    {
        helios::vec3 grid_size = getCellSize(c);
        helios::vec3 grid_center = getCellCenter(c);
        
        file_output << c << "," << grid_center.x << "," << grid_center.y << "," << grid_center.z << "," << grid_size.x << "," << grid_size.y << "," << grid_size.z << "," << voxel_beam_count_tot.at(c) << "," << LA_ref[c]/(grid_size.x*grid_size.y*grid_size.z) << "," << Gtheta_ref[c] << "," << dr_bar_ref[c] << "," << dr_var_ref[c] << "," << P_ref[c] << "," << Gtheta[c] << "," << P_first[c] << "," << P_sequal[c] << "," << P_equal[c] << "," << P_intensity[c] << "," << P_exact[c] << "," << LAD_refcheck_Gref[c] << "," << LAD_first_Gref[c] << "," << LAD_sequal_Gref[c] << "," << LAD_equal_Gref[c] << "," << LAD_intensity_Gref[c] << "," << LAD_exact_Gref[c] << "," << LAD_refcheck[c] << "," << LAD_first[c] << "," << LAD_sequal[c] << "," << LAD_equal[c] << "," << LAD_intensity[c] << "," << LAD_exact[c] << std::endl;
    }
    file_output.close();
    
    if( printmessages ){
        std::cout << "done." << std::endl;
    }
    
}

std::vector<float> LiDARcloud::LAD_inversion(std::vector<float> P, std::vector<float> Gtheta, std::vector<std::vector<float>> dr_array, bool fillAnalytic){
    
    float etol = 5e-5;
    uint maxiter = 100;
    
    int Ncells = getGridCellCount();
    
    float error, eold, aold, tmp;
    
    std::vector<float> LAD(Ncells);  
    LAD.resize(Ncells,0.f);
    
    for( uint v=0; v<Ncells; v++ ){
        
        //initial guesses
        float a = 0.1f;
        float h = 0.01f;
        
        float mean = 0.f;
        uint count=0;
        //loop through dr value for each beam
        for( int j=0; j<dr_array.at(v).size(); j++ ){
            mean += exp(-a*dr_array.at(v).at(j)*Gtheta[v]);
            count++;
        }
        mean /= float(count);      
        error = fabs(mean-P[v])/P[v];
        
        tmp = a;
        a = a + h;
        
        uint iter = 0;
        while( error>etol && iter<maxiter){
            
            aold = tmp;
            eold = error;
            
            float mean = 0.f;
            uint count = 0;
            for( int j=0; j<dr_array.at(v).size(); j++ ){
                mean += exp(-a*dr_array.at(v).at(j)*Gtheta[v]);
                count ++;
            }
            assert( count!=0 );
            mean /= float(count);      
            error = fabs(mean-P[v])/P[v];
            
            tmp = a;
            
            if( error==eold ){
                break;
            }
            
            a = fabs( (aold*error-a*eold)/(error-eold) );
            iter++;
            //  std::cout << "iter = " << iter << ", a = " << a << std::endl;
        }
        
        // check if the inversion failed
        if( iter>=maxiter-1 || a!=a || a>100 ){
            
            if(fillAnalytic)
            {
                
                if( printmessages ){
                    std::cout << "WARNING: LAD inversion failed for volume #" << v << ". Using average dr formulation." << std::endl;
                }
                
                // calculate the mean dr for the cell
                float dr_bar = 0;
                for(uint j=0; j<dr_array.at(v).size(); j++)
                {
                    dr_bar += dr_array.at(v).at(j);
                }
                dr_bar /= float(dr_array.at(v).size());
                
                //do the analytic solution
                a =   -log(P.at(v))/(dr_bar*Gtheta.at(v));
                
            }else{
                
                if( printmessages ){
                    std::cout << "WARNING: LAD inversion failed for volume #" << v << ". setting LAD to 999"  << std::endl;
                }
                
                a = 999.0;
            }
        }
        
        LAD[v] = a;
    }
    
    return LAD;
} 


__global__ void LIDAR_CUDA::insideVolume( const uint Nhits, const float3* d_hit_xyz, const uint Ngridcells, const float3* d_grid_size, const float3* d_grid_center, const float3* d_grid_anchor, const float* d_grid_rotation, int* d_hit_vol ){
  
  uint t = blockIdx.x*blockDim.x+threadIdx.x;

  if( t>=Nhits ){
    return;
  }

  d_hit_vol[t] = -1;

  float3 hit_xyz = d_hit_xyz[t];

  for( int i=0; i<Ngridcells; i++ ){

    float3 center = d_grid_center[i];
    float3 anchor = d_grid_anchor[i];
    float3 size = d_grid_size[i];
    float rotation = d_grid_rotation[i];

    float3 origin = make_float3(0,0,0); 

    float3 hit_xyz_rot = d_rotatePoint(hit_xyz-anchor,0,-rotation) + anchor;

    float3 direction = normalize(hit_xyz_rot-origin);

    float ox = origin.x; float oy = origin.y; float oz = origin.z;
    float dx = direction.x; float dy = direction.y; float dz = direction.z;

    float x0 = center.x - 0.5f*size.x; float x1 = center.x + 0.5f*size.x;
    float y0 = center.y - 0.5f*size.y; float y1 = center.y + 0.5f*size.y;
    float z0 = center.z - 0.5f*size.z; float z1 = center.z + 0.5f*size.z;
	
    float tx_min, ty_min, tz_min;
    float tx_max, ty_max, tz_max; 

    float a = 1.0 / dx;
    if (a >= 0) {
      tx_min = (x0 - ox) * a;
      tx_max = (x1 - ox) * a;
    }
    else {
      tx_min = (x1 - ox) * a;
      tx_max = (x0 - ox) * a;
    }
    
    float b = 1.0 / dy;
    if (b >= 0) {
      ty_min = (y0 - oy) * b;
      ty_max = (y1 - oy) * b;
    }
    else {
      ty_min = (y1 - oy) * b;
      ty_max = (y0 - oy) * b;
    }
    
    float c = 1.0 / dz;
    if (c >= 0) {
      tz_min = (z0 - oz) * c;
      tz_max = (z1 - oz) * c;
    }
    else {
      tz_min = (z1 - oz) * c;
      tz_max = (z0 - oz) * c;
    }
    
    float t0, t1;
    
    // find largest entering t value
    
    if (tx_min > ty_min)
      t0 = tx_min;
    else
      t0 = ty_min;
    
    if (tz_min > t0)
      t0 = tz_min;	
    
    // find smallest exiting t value
    
    if (tx_max < ty_max)
      t1 = tx_max;
    else
      t1 = ty_max;
    
    if (tz_max < t1)
      t1 = tz_max;
    
    if (t0 < t1 && t1 > 1e-6){ //Ray passed through box
      float T = magnitude(hit_xyz_rot-origin);
      if( T>=t0 && T<=t1 ){ //Ray endpoint is inside box
	d_hit_vol[t] = i;
	return;
      }
    }
  
  }

}

__global__ void LIDAR_CUDA::intersectBoundingBox( const size_t scanSize, const float3 origin, const float3* d_scan_xyz, const float3 bbcenter, const float3 bbsize, uint* d_boundingbox_hit ){
  
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;

  if( i>=scanSize ){
    return;
  }

  float3 scan_xyz = d_scan_xyz[ i ];

  float3 direction = normalize( scan_xyz - origin );

  float ox = origin.x; float oy = origin.y; float oz = origin.z;
  float dx = direction.x; float dy = direction.y; float dz = direction.z;

  float x0 = bbcenter.x - 0.5f*bbsize.x; float x1 = bbcenter.x + 0.5f*bbsize.x;
  float y0 = bbcenter.y - 0.5f*bbsize.y; float y1 = bbcenter.y + 0.5f*bbsize.y;
  float z0 = bbcenter.z - 0.5f*bbsize.z; float z1 = bbcenter.z + 0.5f*bbsize.z;

  //first check if we are inside the bounding box

  if( ox>=x0 && ox<=x1 && oy>=y0 && oy<=y1 && oz>=z0 && oz<=z1 ){
    d_boundingbox_hit[ i ] = 1;
    return;
  }

  //if not inside, intersect bounding box
  
  float tx_min, ty_min, tz_min;
  float tx_max, ty_max, tz_max; 

  float a = 1.0 / dx;
  if (a >= 0) {
    tx_min = (x0 - ox) * a;
    tx_max = (x1 - ox) * a;
  }
  else {
    tx_min = (x1 - ox) * a;
    tx_max = (x0 - ox) * a;
  }
  
  float b = 1.0 / dy;
  if (b >= 0) {
    ty_min = (y0 - oy) * b;
    ty_max = (y1 - oy) * b;
  }
  else {
    ty_min = (y1 - oy) * b;
    ty_max = (y0 - oy) * b;
  }
  
  float c = 1.0 / dz;
  if (c >= 0) {
    tz_min = (z0 - oz) * c;
    tz_max = (z1 - oz) * c;
  }
  else {
    tz_min = (z1 - oz) * c;
    tz_max = (z0 - oz) * c;
  }
    
  float t0, t1;
    
  // find largest entering t value
    
  if (tx_min > ty_min)
    t0 = tx_min;
  else
    t0 = ty_min;
    
  if (tz_min > t0)
    t0 = tz_min;	
		
  // find smallest exiting t value
    
  if (tx_max < ty_max)
    t1 = tx_max;
  else
    t1 = ty_max;
  
  if (tz_max < t1)
    t1 = tz_max;

  if (t0 < t1 && t1 > 1e-6){ //Ray passed through box
    d_boundingbox_hit[ i ] = 1;
  }

}

__global__ void LIDAR_CUDA::intersectGridcell( const size_t Nhitsbb, const float3 origin, float3* d_scan_xyz, float* d_scan_weight, const float3 center, const float3 anchor, const float3 size, const float rotation, float* d_dr, float* hit_before, float* hit_after ){
  
  size_t idx = blockIdx.x*blockDim.x+threadIdx.x;

  if( idx>=Nhitsbb ){
    return;
  }

  float3 scan_xyz = d_scan_xyz[ idx ];

  //Inverse rotate the ray

  float3 origin_rot = d_rotatePoint(origin-anchor,0,-rotation) + anchor;
  float3 scan_xyz_rot = d_rotatePoint(scan_xyz-anchor,0,-rotation) + anchor;

  float3 direction = normalize(scan_xyz_rot-origin_rot);

  float ox = origin_rot.x; float oy = origin_rot.y; float oz = origin_rot.z;
  float dx = direction.x; float dy = direction.y; float dz = direction.z;
  
  float x0 = center.x - 0.5f*size.x; float x1 = center.x + 0.5f*size.x;
  float y0 = center.y - 0.5f*size.y; float y1 = center.y + 0.5f*size.y;
  float z0 = center.z - 0.5f*size.z; float z1 = center.z + 0.5f*size.z;
  
  float tx_min, ty_min, tz_min;
  float tx_max, ty_max, tz_max; 
  
  float a = 1.0 / dx;
  if (a >= 0) {
    tx_min = (x0 - ox) * a;
    tx_max = (x1 - ox) * a;
  }
  else {
    tx_min = (x1 - ox) * a;
    tx_max = (x0 - ox) * a;
  }
  
  float b = 1.0 / dy;
  if (b >= 0) {
    ty_min = (y0 - oy) * b;
    ty_max = (y1 - oy) * b;
  }
  else {
    ty_min = (y1 - oy) * b;
    ty_max = (y0 - oy) * b;
  }
  
  float c = 1.0 / dz;
  if (c >= 0) {
    tz_min = (z0 - oz) * c;
    tz_max = (z1 - oz) * c;
  }
  else {
    tz_min = (z1 - oz) * c;
    tz_max = (z0 - oz) * c;
  }
  
  float t0, t1;
    
  // find largest entering t value
  
  if (tx_min > ty_min)
    t0 = tx_min;
  else
    t0 = ty_min;
  
  if (tz_min > t0)
    t0 = tz_min;	
		
  // find smallest exiting t value
    
  if (tx_max < ty_max)
    t1 = tx_max;
  else
    t1 = ty_max;
  
  if (tz_max < t1)
    t1 = tz_max;
  
  if (t0 < t1 && t1 > 1e-6){ //Ray passed through box
    float t = magnitude( scan_xyz_rot - origin_rot );  //t-distance to hit point
    
    if( t>=t0 ){ //hit lies within or beyond the volume

      atomicAdd( hit_after, sin(acos_safe(dz))*d_scan_weight[idx] );

      d_dr[ idx ] = fabs(t1-t0);
	
    }else if( t<t0 ){ //hit lies before the volume

      atomicAdd( hit_before, sin(acos_safe(dz))*d_scan_weight[idx] );
            
    }
    
  }

}

__global__ void LIDAR_CUDA::intersectGridcell_synthetic( const size_t Nhitsbb, const float3 origin, float3* d_scan_xyz, float* d_scan_weight, const float3 center, const float3 anchor, const float3 size, const float rotation, float* d_dr, float* hit_before, float* hit_after, uint* d_hit_location, const float scanner_range ){
    
    size_t idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    if( idx>=Nhitsbb ){
        return;
    }
    
    float3 scan_xyz = d_scan_xyz[ idx ];
    
    //Inverse rotate the ray
    
    float3 origin_rot = d_rotatePoint(origin-anchor,0,-rotation) + anchor;
    float3 scan_xyz_rot = d_rotatePoint(scan_xyz-anchor,0,-rotation) + anchor;
    
    float3 direction = normalize(scan_xyz_rot-origin_rot);
    
    float ox = origin_rot.x; float oy = origin_rot.y; float oz = origin_rot.z;
    float dx = direction.x; float dy = direction.y; float dz = direction.z;
    
    float x0 = center.x - 0.5f*size.x; float x1 = center.x + 0.5f*size.x;
    float y0 = center.y - 0.5f*size.y; float y1 = center.y + 0.5f*size.y;
    float z0 = center.z - 0.5f*size.z; float z1 = center.z + 0.5f*size.z;
    
    float tx_min, ty_min, tz_min;
    float tx_max, ty_max, tz_max; 
    
    float a = 1.0 / dx;
    if (a >= 0) {
        tx_min = (x0 - ox) * a;
        tx_max = (x1 - ox) * a;
    }
    else {
        tx_min = (x1 - ox) * a;
        tx_max = (x0 - ox) * a;
    }
    
    float b = 1.0 / dy;
    if (b >= 0) {
        ty_min = (y0 - oy) * b;
        ty_max = (y1 - oy) * b;
    }
    else {
        ty_min = (y1 - oy) * b;
        ty_max = (y0 - oy) * b;
    }
    
    float c = 1.0 / dz;
    if (c >= 0) {
        tz_min = (z0 - oz) * c;
        tz_max = (z1 - oz) * c;
    }
    else {
        tz_min = (z1 - oz) * c;
        tz_max = (z0 - oz) * c;
    }
    
    float t0, t1;
    
    // find largest entering t value
    
    if (tx_min > ty_min)
        t0 = tx_min;
    else
        t0 = ty_min;
    
    if (tz_min > t0)
        t0 = tz_min;	
    
    // find smallest exiting t value
    
    if (tx_max < ty_max)
        t1 = tx_max;
    else
        t1 = ty_max;
    
    if (tz_max < t1)
        t1 = tz_max;
    
    if (t0 < t1 && t1 > 1e-6){ //Ray passed through box
        float t = magnitude( scan_xyz_rot - origin_rot );  //t-distance to hit point
        
        if( t>=t0 && t<= t1 ){ //hit lies within the volume
            atomicAdd( hit_after, sin(acos_safe(dz))*d_scan_weight[idx] );
            d_hit_location[ idx ] = 2;        
            d_dr[ idx ] = fabs(t1-t0);
        }else if( t > t1 && t < scanner_range){ //hit lies after the volume (but not a miss)
            d_hit_location[ idx ] = 3;
            d_dr[ idx ] = fabs(t1-t0);
        }else if( t > t1 && t >= scanner_range){ //hit lies after volume but out of range of scanner (miss)
            d_hit_location[ idx ] = 4;
            d_dr[ idx ] = fabs(t1-t0);
        }else if( t<t0 ){ //hit lies before the volume
            d_hit_location[ idx ] = 1;
            atomicAdd( hit_before, sin(acos_safe(dz))*d_scan_weight[idx] );
            
        }
        
    }
    
}

void LiDARcloud::trunkReconstruction( const helios::vec3 box_center, const helios::vec3 box_size, const float Lmax, const float max_aspect_ratio ){

  if( printmessages ){
    std::cout << "Performing trunk reconstruction..." << std::flush;
  }
    
  // ---- Determine which points are in trunk volume ---- //

  float3* d_hit_xyz;

  const uint N = getHitCount();
  
  float3* hit_xyz = (float3*)malloc(N * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_xyz,N*sizeof(float3)) ); //allocate device memory

  //copy scan data into the host buffer
  for( std::size_t r=0; r<N; r++ ){
    hit_xyz[r] = vec3tofloat3(getHitXYZ(r));
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_hit_xyz, hit_xyz, N*sizeof(float3), hipMemcpyHostToDevice) );

  const uint Ncells = getGridCellCount();

  float3 grid_center = vec3tofloat3(box_center);
  float3 grid_size = vec3tofloat3(box_size);
  float grid_rotation = 0;
  
  float3* d_grid_center;
  float3* d_grid_size;
  float* d_grid_rotation;

  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_center,sizeof(float3)) );
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_size,sizeof(float3)) );
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_rotation,sizeof(float)) );

  CUDA_CHECK_ERROR( hipMemcpy(d_grid_center, &grid_center, sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_size, &grid_size, sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_rotation, &grid_rotation, sizeof(float), hipMemcpyHostToDevice) );
  
  free(hit_xyz);

  // Result buffer
  int* hit_vol = (int*)malloc( N * sizeof(int));
  int* d_hit_vol;
  CUDA_CHECK_ERROR( hipMalloc(&d_hit_vol,N*sizeof(int)) );

  dim3 dimBlock( 64, 1 );
  dim3 dimGrid( ceil(N/64.f) );
  insideVolume <<< dimGrid, dimBlock >>>( N, d_hit_xyz, 1, d_grid_size, d_grid_center, d_grid_center, d_grid_rotation, d_hit_vol );

  CUDA_CHECK_ERROR( hipPeekAtLastError() );
  CUDA_CHECK_ERROR( hipDeviceSynchronize() );
  
  CUDA_CHECK_ERROR( hipMemcpy(hit_vol, d_hit_vol, N*sizeof(int), hipMemcpyDeviceToHost) );

  CUDA_CHECK_ERROR( hipFree(d_hit_vol) );
  CUDA_CHECK_ERROR( hipFree(d_hit_xyz) );
  CUDA_CHECK_ERROR( hipFree(d_grid_center) );
  CUDA_CHECK_ERROR( hipFree(d_grid_size) );
  CUDA_CHECK_ERROR( hipFree(d_grid_rotation) );

  // ------ Triangulate trunk points -------- //

  for( uint s=0; s<getScanCount(); s++ ){
    
    std::vector<int> Delaunay_inds;

    std::vector<Shx> pts;
    Shx pt;
    
    int count = 0;
    for( int r=0; r<N; r++ ){

      if( getHitScanID(r)==s && hit_vol[r]>=0 ){
      
  	helios::SphericalCoord direction = getHitRaydir(r);
	
  	//pt.id = count;
	pt.id = r;
  	pt.r = direction.zenith;
  	pt.c = direction.azimuth;

  	pts.push_back(pt);

  	Delaunay_inds.push_back(r);

  	count++;

      }
	
    }

    std::vector<int> dupes;
    int nx = de_duplicate( pts, dupes);

    std::vector<Triad> triads;

    s_hull_pro( pts, triads );

    std::ofstream file;
    file.open("nodes.txt");

    std::ofstream filea;
    filea.open("aspect.txt");

    std::ofstream filev;
    filev.open("vertices.txt");
    for( int r=0; r<N; r++ ){
      helios::vec3 pos = getHitXYZ(r);
      filev << pos.x << " " << pos.y << " " << pos.z << std::endl;
    }
    filev.close();

    std::ofstream filed;
    filed.open("angles.txt");
    for( int r=0; r<N; r++ ){
      if( getHitScanID(r)==s && hit_vol[r]>=0 ){
	helios::SphericalCoord direction = getHitRaydir(r);
	filed << direction.zenith << " " << direction.azimuth << std::endl;
      }
    }
    filed.close();

    std::vector<Triangulation> trunk_triangles;
    for( int t=0; t<triads.size(); t++ ){

      // int ID0 = Delaunay_inds.at(triads.at(t).a);
      // int ID1 = Delaunay_inds.at(triads.at(t).b);
      // int ID2 = Delaunay_inds.at(triads.at(t).c);
      int ID0 = triads.at(t).a;
      int ID1 = triads.at(t).b;
      int ID2 = triads.at(t).c;
      
      helios::vec3 vertex0 = getHitXYZ( ID0 );
      helios::vec3 vertex1 = getHitXYZ( ID1 );
      helios::vec3 vertex2 = getHitXYZ( ID2 );

      helios::vec3 v;
      v=vertex0-vertex1;
      float L0 = v.magnitude();
      v=vertex0-vertex2;
      float L1 = v.magnitude();
      v=vertex1-vertex2;
      float L2 = v.magnitude();
      
      float aspect_ratio = fmax(fmax(L0,L1),L2)/fmin(fmin(L0,L1),L2);

      if( L0>Lmax || L1>Lmax || L2>Lmax || aspect_ratio>max_aspect_ratio ){
  	continue;
      }
	    
      helios::RGBcolor color = helios::make_RGBcolor(0,0,0);
      color.r = (hits.at(ID0).color.r + hits.at(ID1).color.r + hits.at(ID2).color.r )/3.f;
      color.g = (hits.at(ID0).color.g + hits.at(ID1).color.g + hits.at(ID2).color.g )/3.f;
      color.b = (hits.at(ID0).color.b + hits.at(ID1).color.b + hits.at(ID2).color.b )/3.f;
  
      Triangulation tri( s, vertex0, vertex1, vertex2, ID0, ID1, ID2, color, 0 );

      trunk_triangles.push_back(tri);

      file << ID0 << " " << ID1 << " " << ID2 << std::endl;
      filea << aspect_ratio << std::endl;

    }

    file.close();
    filea.close();

    // ------ Flood fill -------- //

    int current_group = 0;

    std::vector<std::vector<int> > nodes;
    nodes.resize(getHitCount());

    size_t Ntri = trunk_triangles.size();
    
    for( size_t t=0; t<Ntri; t++ ){

      Triangulation tri = trunk_triangles.at(t);

      nodes.at( tri.ID0 ).push_back(t);
      nodes.at( tri.ID1 ).push_back(t);
      nodes.at( tri.ID2 ).push_back(t);

    }

    std::vector<int> fill_flag;
    fill_flag.resize(Ntri,-1);
    
    for( size_t t=0; t<Ntri; t++ ){//looping through all triangles

      if( fill_flag.at(t)<0 ){

	floodfill( t, trunk_triangles, fill_flag, nodes, current_group, 0, 1e6 );

	current_group ++;

      }
      
    }

    std::vector<std::vector<Triangulation> > trunk_groups;

    for( size_t t=0; t<Ntri; t++ ){//looping through all triangles

      if( fill_flag.at(t)>=0 ){
	int fill_group = fill_flag.at(t);

	if( fill_group>=trunk_groups.size() ){
	  trunk_groups.resize( fill_group+1 );
	}
	
	trunk_groups.at(fill_group).push_back(trunk_triangles.at(t));

      }

    }

    //calculate the area of fill groups

    float max_area = 0.f;
    int max_ind = 0;
    for( size_t i=0; i<trunk_groups.size(); i++ ){
      
      float group_area = 0;
      for( size_t t=0; t<trunk_groups.at(i).size(); t++ ){
	float area = trunk_groups.at(i).at(t).area;
	if( area==area ){
	  group_area += area;
	}
      }

      if( group_area>max_area ){
    	max_area = group_area;
    	max_ind = i;
      }
      
    }

    reconstructed_trunk_triangles.push_back(trunk_groups.at(max_ind));
  
  }
    
  free(hit_vol);

  if( printmessages ){
    std::cout << "done." << std::endl;
  }

}

void LiDARcloud::syntheticScan( helios::Context* context ){
  syntheticScan( context, 1, 0, false, false );
}

void LiDARcloud::syntheticScan( helios::Context* context, const bool scan_grid_only, const bool record_misses ){
  syntheticScan( context, 1, 0, scan_grid_only, record_misses );
}

void LiDARcloud::syntheticScan( helios::Context* context, const int rays_per_pulse, const float pulse_distance_threshold ){
  syntheticScan( context, rays_per_pulse, pulse_distance_threshold, false, false );
}

void LiDARcloud::syntheticScan( helios::Context* context, const int rays_per_pulse, const float pulse_distance_threshold, const bool scan_grid_only, const bool record_misses ){

  int Npulse;
  if( rays_per_pulse<1 ){
    Npulse=1;
  }else{
    Npulse=rays_per_pulse;
  }

  if( printmessages ){
    if( Npulse>1 ){
      std::cout << "Performing full-waveform synthetic LiDAR scan..." << std::endl;
    }else{
      std::cout << "Performing discrete return synthetic LiDAR scan..." << std::endl;
    }
  }

  if( getScanCount()==0 ){
    std::cout << "WARNING (syntheticScan): No scans added to the point cloud. Exiting.." << std::endl;
    return;
  }

  float3 bb_center;
  float3 bb_size;
    
  if(scan_grid_only == false){
    
    //Determine bounding box for Context geometry
    helios::vec2 xbounds, ybounds, zbounds;
    context->getDomainBoundingBox(xbounds,ybounds,zbounds);
    bb_center = make_float3(xbounds.x+0.5*(xbounds.y-xbounds.x),ybounds.x+0.5*(ybounds.y-ybounds.x),zbounds.x+0.5*(zbounds.y-zbounds.x));
    bb_size = make_float3(xbounds.y-xbounds.x,ybounds.y-ybounds.x,zbounds.y-zbounds.x);
    
  }else{
    
    // Determine bounding box for voxels instead of whole domain
    helios::vec3 boxmin, boxmax;
    getGridBoundingBox(boxmin, boxmax);  
    bb_center = make_float3(boxmin.x + 0.5*(boxmax.x-boxmin.x),boxmin.y + 0.5*(boxmax.y-boxmin.y),boxmin.z + 0.5*(boxmax.z-boxmin.z) );
    bb_size = make_float3(boxmax.x-boxmin.x, boxmax.y-boxmin.y, boxmax.z-boxmin.z );
    
  }
  
  //get geometry information and copy to GPU

  size_t c=0;

  std::map<std::string,int> textures;
  std::map<std::string,int2> texture_size;
  std::map<std::string,std::vector<std::vector<bool> > > texture_data;
  int tID = 0;

  std::vector<uint> UUIDs_all = context->getAllUUIDs();

  std::vector<uint> ID_mapping;

  //----- PATCHES ----- //

  //figure out how many patches
  size_t Npatches = 0;
  for( int p=0; p<UUIDs_all.size(); p++ ){
    if( context->getPrimitiveType(UUIDs_all.at(p)) == helios::PRIMITIVE_TYPE_PATCH ){
      Npatches++;
    }
  }

  ID_mapping.resize(Npatches);

  float3* patch_vertex = (float3*)malloc(4*Npatches * sizeof(float3)); //allocate host memory
  int* patch_textureID = (int*)malloc(Npatches * sizeof(int)); //allocate host memory
  float2* patch_uv = (float2*)malloc(2*Npatches * sizeof(float2)); //allocate host memory

  c=0;
  for( int p=0; p<UUIDs_all.size(); p++ ){
      uint UUID = UUIDs_all.at(p);
      if( context->getPrimitiveType(UUID) == helios::PRIMITIVE_TYPE_PATCH ){
          std::vector<helios::vec3> verts = context->getPrimitiveVertices(UUID);
          patch_vertex[4*c] = vec3tofloat3(verts.at(0));
          patch_vertex[4*c+1] = vec3tofloat3(verts.at(1));
          patch_vertex[4*c+2] = vec3tofloat3(verts.at(2));
          patch_vertex[4*c+3] = vec3tofloat3(verts.at(3));

          ID_mapping.at(c) = UUIDs_all.at(p);

          if( !context->getPrimitiveTextureFile(UUID).empty() && context->primitiveTextureHasTransparencyChannel(UUID) ){
              std::string tex = context->getPrimitiveTextureFile(UUID);
              std::map<std::string,int>::iterator it = textures.find(tex);
              if( it != textures.end() ){ //texture already exits
                  patch_textureID[c] = textures.at(tex);
              }else{ //new texture
                  patch_textureID[c] = tID;
                  textures[tex] = tID;
                  helios::int2 tsize = context->getPrimitiveTextureSize(UUID);
                  texture_size[tex] = make_int2(tsize.x,tsize.y);
                  texture_data[tex] = *context->getPrimitiveTextureTransparencyData(UUID);
                  tID++;
              }

              std::vector<helios::vec2> uv = context->getPrimitiveTextureUV(UUID);
              if( uv.size()==4 ){//cusom uv coordinates
                  patch_uv[2*c] = vec2tofloat2(uv.at(1));
                  patch_uv[2*c+1] = vec2tofloat2(uv.at(3));
              }else{//default uv coordinates
                  patch_uv[2*c] = make_float2(0,0);
                  patch_uv[2*c+1] = make_float2(1,1);
              }

          }else{
              patch_textureID[c]=-1;
          }

          c++;
      }
  }
  
  float3* d_patch_vertex;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_patch_vertex,4*Npatches*sizeof(float3)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_patch_vertex, patch_vertex, 4*Npatches*sizeof(float3), hipMemcpyHostToDevice) );
  int* d_patch_textureID;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_patch_textureID,Npatches*sizeof(int)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_patch_textureID, patch_textureID, Npatches*sizeof(int), hipMemcpyHostToDevice) );
  float2* d_patch_uv;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_patch_uv, 2*Npatches*sizeof(float2)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_patch_uv, patch_uv, 2*Npatches*sizeof(float2), hipMemcpyHostToDevice) );

  //----- TRIANGLES ----- //

  //figure out how many triangles
  size_t Ntriangles = 0;
  for( int p=0; p<UUIDs_all.size(); p++ ){
    if( context->getPrimitiveType(UUIDs_all.at(p)) == helios::PRIMITIVE_TYPE_TRIANGLE ){
      Ntriangles++;
    }
  }

  ID_mapping.resize(Npatches+Ntriangles);

  float3* tri_vertex = (float3*)malloc(3*Ntriangles * sizeof(float3)); //allocate host memory
  int* tri_textureID = (int*)malloc(Ntriangles * sizeof(int)); //allocate host memory
  float2* tri_uv = (float2*)malloc(3*Ntriangles * sizeof(float2)); //allocate host memory

  c=0;
  for( int p=0; p<UUIDs_all.size(); p++ ){
      uint UUID = UUIDs_all.at(p);
      if( context->getPrimitiveType(UUID) == helios::PRIMITIVE_TYPE_TRIANGLE ){
          std::vector<helios::vec3> verts = context->getPrimitiveVertices(UUID);
          tri_vertex[3*c] = vec3tofloat3(verts.at(0));
          tri_vertex[3*c+1] = vec3tofloat3(verts.at(1));
          tri_vertex[3*c+2] = vec3tofloat3(verts.at(2));

          ID_mapping.at(Npatches+c) = UUIDs_all.at(p);

          if( !context->getPrimitiveTextureFile(UUID).empty() && context->primitiveTextureHasTransparencyChannel(UUID) ){
              std::string tex = context->getPrimitiveTextureFile(UUID);
              std::map<std::string,int>::iterator it = textures.find(tex);
              if( it != textures.end() ){ //texture already exits
                  tri_textureID[c] = textures.at(tex);
              }else{ //new texture
                  tri_textureID[c] = tID;
                  textures[tex] = tID;
                  helios::int2 tsize = context->getPrimitiveTextureSize(UUID);
                  texture_size[tex] = make_int2(tsize.x,tsize.y);
                  texture_data[tex] = *context->getPrimitiveTextureTransparencyData(UUID);
                  tID++;
              }

              std::vector<helios::vec2> uv = context->getPrimitiveTextureUV(UUID);
              assert( uv.size()==3 );
              tri_uv[3*c] = vec2tofloat2(uv.at(0));
              tri_uv[3*c+1] = vec2tofloat2(uv.at(1));
              tri_uv[3*c+2] = vec2tofloat2(uv.at(2));

          }else{
              tri_textureID[c]=-1;
          }

          c++;
      }
  }

  float3* d_tri_vertex;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_tri_vertex,3*Ntriangles*sizeof(float3)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_tri_vertex, tri_vertex, 3*Ntriangles*sizeof(float3), hipMemcpyHostToDevice) );
  int* d_tri_textureID;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_tri_textureID, Ntriangles*sizeof(int)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_tri_textureID, tri_textureID, Ntriangles*sizeof(int), hipMemcpyHostToDevice) );
  float2* d_tri_uv;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_tri_uv,3*Ntriangles*sizeof(float2)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_tri_uv, tri_uv, 3*Ntriangles*sizeof(float2), hipMemcpyHostToDevice) );
  
  //transfer texture data to GPU
  const int Ntextures = textures.size();

  int2 masksize_max = make_int2(0,0);
  for( std::map<std::string,int2>::iterator it=texture_size.begin(); it!=texture_size.end(); ++it ){
    if( it->second.x>masksize_max.x ){
      masksize_max.x=it->second.x;
    }
    if( it->second.y>masksize_max.y ){
      masksize_max.y=it->second.y;
    }
  }

  bool* maskdata = (bool*)malloc(Ntextures*masksize_max.x*masksize_max.y * sizeof(bool)); //allocate host memory
  int2* masksize = (int2*)malloc(Ntextures * sizeof(int2)); //allocate host memory

  for( std::map<std::string,int2>::iterator it=texture_size.begin(); it!=texture_size.end(); ++it ){
    std::string texture_file = it->first;

    int ID = textures.at(texture_file); 
    
    masksize[ID] = it->second;

    int ind=0;
    for( int j=0; j<masksize_max.y; j++ ){
      for( int i=0; i<masksize_max.x; i++ ){

	if( i<texture_size.at(texture_file).x && j<texture_size.at(texture_file).y ){
	  maskdata[ID*masksize_max.x*masksize_max.y+ind] = texture_data.at(texture_file).at(j).at(i);
	}else{
	  maskdata[ID*masksize_max.x*masksize_max.y+ind] = false;
	}
	ind++;
      }
    }
  }

  bool* d_maskdata;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_maskdata, Ntextures*masksize_max.x*masksize_max.y * sizeof(bool)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_maskdata, maskdata, Ntextures*masksize_max.x*masksize_max.y * sizeof(bool), hipMemcpyHostToDevice) );
  int2* d_masksize;
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_masksize, Ntextures * sizeof(int2)) ); //allocate device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_masksize, masksize, Ntextures * sizeof(int2), hipMemcpyHostToDevice) );

  for( int s=0; s<getScanCount(); s++ ){

    float3 scan_origin = vec3tofloat3(getScanOrigin(s));

    int Ntheta = getScanSizeTheta(s);
    int Nphi = getScanSizePhi(s);

    helios::vec2 thetarange = getScanRangeTheta(s);
    float thetamin = thetarange.x;
    float thetamax = thetarange.y;
    helios::vec2 phirange = getScanRangePhi(s);
    float phimin = phirange.x;
    float phimax = phirange.y;

    std::vector<std::string> column_format = getScanColumnFormat(s);
    
    std::vector<helios::vec3> raydir;
    raydir.resize(Ntheta*Nphi);
  
    for (uint j=0; j<Nphi; j++ ){
      float phi = phimin+float(j)*(phimax-phimin)/float(Nphi);
      for (uint i=0; i<Ntheta; i++ ){        
	float theta_z = thetamin+float(i)*(thetamax-thetamin)/float(Ntheta);
	float theta_elev = 0.5f*M_PI-theta_z;
	helios::vec3 dir = sphere2cart(helios::make_SphericalCoord(1.f,theta_elev,phi));
	raydir.at(Ntheta*j+i) = dir;
      }
    }

    size_t N = Ntheta*Nphi;

    float3* d_hit_xyz;
    CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_xyz,N*sizeof(float3)) ); //allocate device memory

    //copy scan data into the host buffer
    float3* hit_xyz = (float3*)malloc(N * sizeof(float3)); //allocate host memory
    for( std::size_t r=0; r<N; r++ ){
      hit_xyz[r] = scan_origin+vec3tofloat3(raydir.at(r)*10000.f);
    }

    //copy from host to device memory
    CUDA_CHECK_ERROR( hipMemcpy(d_hit_xyz, hit_xyz, N*sizeof(float3), hipMemcpyHostToDevice) );

    uint* bb_hit = (uint*)malloc(N * sizeof(uint)); //allocate host memory
    uint* d_bb_hit;
    CUDA_CHECK_ERROR( hipMalloc((void**)&d_bb_hit,N*sizeof(uint)) ); //allocate device memory
    CUDA_CHECK_ERROR( hipMemset( d_bb_hit, 0, N*sizeof(uint)) ); //initialize to zero, set equal to 1 if the ray is found to intersect bounding box
    
    //Launch kernel to determine which rays intersect bounding box
    uint3 dimBlock = make_uint3( 512, 1, 1 );
    uint3 dimGrid = make_uint3( ceil(float(N)/float(dimBlock.x)), 1, 1 );
    intersectBoundingBox<<< dimGrid, dimBlock >>>( N, scan_origin, d_hit_xyz, bb_center, bb_size, d_bb_hit );

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here

    //copy hit flag back to host
    CUDA_CHECK_ERROR( hipMemcpy(bb_hit, d_bb_hit, N*sizeof(uint), hipMemcpyDeviceToHost) );

    CUDA_CHECK_ERROR( hipFree(d_hit_xyz) );
    CUDA_CHECK_ERROR( hipFree(d_bb_hit) );

    //determine how many rays hit the bounding box
    N = 0;
    float hit_out = 0;
    for( int i=0; i<Ntheta*Nphi; i++ ){
      if( bb_hit[i]==1 ){
	N++;
	helios::SphericalCoord dir = cart2sphere(raydir[i]);
	hit_out += sin(dir.zenith);
      }
    }

    if( N==0 ){
      std::cout << "WARNING: Synthetic rays did not hit any primitives." << std::endl;
      return;
    }

    //make a new array of ray directions for rays that hit bounding box
    float3* direction = (float3*)malloc(N * sizeof(float3)); //allocate host memory

    int count=0;
    for( int i=0; i<Ntheta*Nphi; i++ ){
      if( bb_hit[i]==1 ){
	direction[count] = vec3tofloat3(raydir.at(i));
	count++;
      }
    }    
    free(bb_hit);

    float3* d_raydir;
    CUDA_CHECK_ERROR( hipMalloc((void**)&d_raydir,N*sizeof(float3)) ); //allocate device memory
    CUDA_CHECK_ERROR( hipMemcpy(d_raydir, direction, N*sizeof(float3), hipMemcpyHostToDevice) );

    //Distance to intersection
    float* d_hit_t;
    CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_t,N*Npulse*sizeof(float)) ); //allocate device memory
    float* hit_t = (float*)malloc(N*Npulse * sizeof(float)); //allocate host memory
    for( int i=0; i<N*Npulse; i++ ){
      hit_t[i] = 1e5;
    }
    CUDA_CHECK_ERROR( hipMemcpy(d_hit_t, hit_t, N*Npulse*sizeof(float), hipMemcpyHostToDevice) );

    //Dot product of primitive normal and ray direction (for calculating intensity)
    float* d_hit_fnorm;
    CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_fnorm,N*Npulse*sizeof(float)) ); //allocate device memory
    float* hit_fnorm = (float*)malloc(N*Npulse * sizeof(float)); //allocate host memory
    for( int i=0; i<N*Npulse; i++ ){
      hit_fnorm[i] = 1e6;
    }
    CUDA_CHECK_ERROR( hipMemcpy(d_hit_fnorm, hit_fnorm, N*Npulse*sizeof(float), hipMemcpyHostToDevice) );

    //Dot product of primitive normal and ray direction (for calculating intensity)
    int* d_hit_ID;
    CUDA_CHECK_ERROR( hipMalloc((void**)&d_hit_ID,N*Npulse*sizeof(int)) ); //allocate device memory
    int* hit_ID = (int*)malloc(N*Npulse * sizeof(int)); //allocate host memory
    for( int i=0; i<N*Npulse; i++ ){
      hit_ID[i] = 999999999;
    }
    CUDA_CHECK_ERROR( hipMemcpy(d_hit_ID, hit_ID, N*Npulse*sizeof(float), hipMemcpyHostToDevice) );

    float exit_diameter = getScanBeamExitDiameter(s);
    float beam_divergence = getScanBeamDivergence(s);

      if( Npulse>1 ){
          dimBlock = make_uint3( 128, 4, 1 );
      }else{
          dimBlock = make_uint3( 512, 1, 1 );
      }
    dimGrid = make_uint3( ceil(float(N)/float(dimBlock.x)), ceil(float(Npulse)/float(dimBlock.y)), 1 );

    //---- patch kernel ----//
    intersectPatches<<< dimGrid, dimBlock >>>( N, Npulse, scan_origin, d_raydir, exit_diameter, beam_divergence, Npatches, d_patch_vertex, d_patch_textureID, Ntextures, d_masksize, masksize_max, d_maskdata, d_patch_uv, d_hit_t, d_hit_fnorm, d_hit_ID );

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here
      
    //---- triangle kernel ----//
    intersectTriangles<<< dimGrid, dimBlock >>>( N, Npulse, scan_origin, d_raydir, exit_diameter, beam_divergence, Ntriangles, Npatches, d_tri_vertex, d_tri_textureID, Ntextures, d_masksize, masksize_max, d_maskdata, d_tri_uv, d_hit_t, d_hit_fnorm, d_hit_ID );
    
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR( hipPeekAtLastError() ); //if there was an error inside the kernel, it will show up here
    
    //copy back
    CUDA_CHECK_ERROR( hipMemcpy(hit_t, d_hit_t, N*Npulse*sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK_ERROR( hipMemcpy(hit_fnorm, d_hit_fnorm, N*Npulse*sizeof(float), hipMemcpyDeviceToHost) );
    CUDA_CHECK_ERROR( hipMemcpy(hit_ID, d_hit_ID, N*Npulse*sizeof(int), hipMemcpyDeviceToHost) );

    size_t Nhits = 0;
    //looping over beams
    for( size_t r=0; r<N; r++ ){

      std::vector<std::vector<float> > t_pulse;
      std::vector<std::vector<float> > t_hit;

      //looping over rays in each beam
      for( size_t p=0; p<Npulse; p++ ){
	
    	float t = hit_t[r*Npulse+p];  //distance to hit (misses t=1e6)
	float i = hit_fnorm[r*Npulse+p]; //dot product between beam direction and primitive normal
	float ID = float(hit_ID[r*Npulse+p]);   //ID of intersected primitive

    	if( record_misses || (!record_misses && t<1e5) ){ 
	  std::vector<float> v{t,i,ID};
	  t_pulse.push_back(v);
	}

      }

      if( t_pulse.size()==1 ){ //this is discrete-return data, or we only had one hit for this pulse

	std::vector<float> v{t_pulse.front().at(0),t_pulse.front().at(1),1,t_pulse.front().at(2)};
    	t_hit.push_back( v );

      }else if( t_pulse.size()>1 ){ //more than one hit for this pulse
	
    	std::sort( t_pulse.begin(), t_pulse.end(), LIDAR_CUDA::sortcol0 );

     	float t0=t_pulse.at(0).at(0);
    	float d=t_pulse.at(0).at(0);
	float f=t_pulse.at(0).at(1);
    	int count=1;

	//loop over rays in each beam and group into hit points
	for( size_t hit=1; hit<=t_pulse.size(); hit++ ){
                    
	  // if the end has been reached, output the last hitpoint
	  if( hit == t_pulse.size()){
                        
	    std::vector<float> v{d/float(count), f/float(Npulse), float(count), t_pulse.at(hit-1).at(2) }; //Note: the last index of t_pulse (.at(2)) is the object identifier. We don't want object identifiers to be averaged, so we'll assign the hit identifier based on the last ray in the group
	    t_hit.push_back( v );
                        
	    // else if the current ray is more than the pulse threshold distance from t0,  it is part of the next hitpoint so output the previous hitpoint and reset
	  }else if( t_pulse.at(hit).at(0)-t0>pulse_distance_threshold ){
                        
	    std::vector<float> v{d/float(count), f/float(Npulse), float(count), t_pulse.at(hit-1).at(2)}; //included the ray count here
	    //Note: the last index of t_pulse (.at(2)) is the object identifier. We don't want object identifiers to be averaged, so we'll assign the hit identifier based on the last ray in the group
	    t_hit.push_back( v );
            
	    Nhits++;
	    count=1;
	    d=t_pulse.at(hit).at(0);
	    t0=t_pulse.at(hit).at(0);
	    f=t_pulse.at(hit).at(1);
            
	   // or else the current ray is less than pulse threshold and is part of current hitpoint; add it to the current hit point and continue on
	  }else{
	    
	    count++;
        t0=t_pulse.at(hit).at(0);
	    d+=t_pulse.at(hit).at(0);
	    f+=t_pulse.at(hit).at(1);
            
	  }
          
	}

      }

      float average=0;
      for( size_t hit=0; hit<t_hit.size(); hit++ ){
	average+=t_hit.at(hit).at(0)/float(t_hit.size());
      }

      for( size_t hit=0; hit<t_hit.size(); hit++ ){
	
    	std::map<std::string,double> data;
    	data["target_index"] = hit;
    	data["target_count"] = t_hit.size();
	data["deviation"] = fabs(t_hit.at(hit).at(0)-average);
	data["timestamp"] = r;
	data["intensity"] = t_hit.at(hit).at(1);
	data["distance"] = t_hit.at(hit).at(0);
	data["nRaysHit"] = t_hit.at(hit).at(2);

	float UUID = t_hit.at(hit).at(3);
	if( UUID>=0 && UUID<ID_mapping.size() ){
	  UUID = ID_mapping.at(int(t_hit.at(hit).at(3)));
	}

	helios::RGBcolor color = helios::RGB::red;
	
	if( UUID>=0 && context->doesPrimitiveExist(uint(UUID)) ){
	  
	  color = context->getPrimitiveColor(UUID);
	  
	  if( context->doesPrimitiveDataExist(uint(UUID),"object_label") && context->getPrimitiveDataType(uint(UUID),"object_label")==helios::HELIOS_TYPE_INT ){
	    int label;
	    context->getPrimitiveData(uint(UUID),"object_label",label);
	    data["object_label"] = double(label);
	  }
	}

    	helios::vec3 dir = helios::make_vec3(direction[r].x,direction[r].y,direction[r].z);
    	helios::vec3 origin = helios::make_vec3(scan_origin.x,scan_origin.y,scan_origin.z);
    	helios::vec3 p = origin+dir*t_hit.at(hit).at(0);
    	addHitPoint( s, p, helios::cart2sphere(dir), color, data );

	Nhits++;
      }

    }

    CUDA_CHECK_ERROR( hipFree(d_hit_t) );
    CUDA_CHECK_ERROR( hipFree(d_hit_fnorm) );
    CUDA_CHECK_ERROR( hipFree(d_hit_ID) );
    CUDA_CHECK_ERROR( hipFree(d_raydir) );
    free(hit_xyz);
    free(direction);
    free(hit_t);
    free(hit_fnorm);
    free(hit_ID);

    if( printmessages ){
      std::cout << "Created synthetic scan #" << s << " with " << Nhits << " hit points." << std::endl;
    }
    
  }

  CUDA_CHECK_ERROR( hipFree(d_patch_vertex) );
  CUDA_CHECK_ERROR( hipFree(d_patch_textureID) );
  CUDA_CHECK_ERROR( hipFree(d_patch_uv) );
  CUDA_CHECK_ERROR( hipFree(d_tri_vertex) );
  CUDA_CHECK_ERROR( hipFree(d_tri_textureID) );
  CUDA_CHECK_ERROR( hipFree(d_tri_uv) );
  CUDA_CHECK_ERROR( hipFree(d_maskdata) );
  CUDA_CHECK_ERROR( hipFree(d_masksize) );
  free(patch_vertex);
  free(patch_textureID);
  free(patch_uv);
  free(tri_vertex);
  free(tri_textureID);
  free(tri_uv);
  free(maskdata);
  free(masksize);

  if( printmessages ){
    std::cout << "done." << std::endl;
  }
  
}

std::vector<float> LiDARcloud::calculateSyntheticLeafArea( helios::Context* context ){

  float3* d_prim_xyz;

  std::vector<uint> UUIDs_all = context->getAllUUIDs();

  const uint N = UUIDs_all.size();
  
  float3* prim_xyz = (float3*)malloc(N * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_prim_xyz,N*sizeof(float3)) ); //allocate device memory

  //copy scan data into the host buffer
  for( std::size_t p=0; p<N; p++ ){
    std::vector<helios::vec3> verts = context->getPrimitiveVertices(UUIDs_all.at(p));
    prim_xyz[p] = vec3tofloat3( verts.at(0) );
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_prim_xyz, prim_xyz, N*sizeof(float3), hipMemcpyHostToDevice) );

  // ---- Grid Cells ---- //

  //! Grid cell centers on device (GPU) memory
  float3* d_grid_center;

  //! Grid cell global anchor on device (GPU) memory
  float3* d_grid_anchor;

  //! Grid sizes on device (GPU) memory
  float3* d_grid_size;

  //! Grid rotations on device (GPU) memory
  float* d_grid_rotation;

  const uint Ncells = getGridCellCount();

  float3* center = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_center,Ncells*sizeof(float3)) ); //allocate device memory

  float3* anchor = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_anchor,Ncells*sizeof(float3)) ); //allocate device memory

  float3* size = (float3*)malloc(Ncells * sizeof(float3)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_size,Ncells*sizeof(float3)) ); //allocate device memory

  float* rotation = (float*)malloc(Ncells * sizeof(float)); //allocate host memory
  CUDA_CHECK_ERROR( hipMalloc((void**)&d_grid_rotation,Ncells*sizeof(float)) ); //allocate device memory
  
  //copy grid data into the host buffer
  for( int c=0; c<Ncells; c++ ){
    center[c] = vec3tofloat3(getCellCenter(c));
    anchor[c] = vec3tofloat3(getCellGlobalAnchor(c));
    size[c] = vec3tofloat3(getCellSize(c));
    rotation[c] = getCellRotation(c);
  }

  //copy from host to device memory
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_center, center, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_anchor, anchor, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_size, size, Ncells*sizeof(float3), hipMemcpyHostToDevice) );
  CUDA_CHECK_ERROR( hipMemcpy(d_grid_rotation, rotation, Ncells*sizeof(float), hipMemcpyHostToDevice) );

  free(prim_xyz);
  free(center);
  free(anchor);
  free(size);
  free(rotation);

  // Result buffer
  int* prim_vol = (int*)malloc(N * sizeof(int));
  int* d_prim_vol;
  CUDA_CHECK_ERROR( hipMalloc(&d_prim_vol,N*sizeof(int)) );

  dim3 dimBlock( 64, 1 );
  dim3 dimGrid( ceil(N/64.f) );
  insideVolume <<< dimGrid, dimBlock >>>( N, d_prim_xyz, getGridCellCount(), d_grid_size, d_grid_center, d_grid_anchor, d_grid_rotation, d_prim_vol );

  CUDA_CHECK_ERROR( hipPeekAtLastError() );
  CUDA_CHECK_ERROR( hipDeviceSynchronize() );
  
  CUDA_CHECK_ERROR( hipMemcpy(prim_vol, d_prim_vol, N*sizeof(int), hipMemcpyDeviceToHost) );

  std::vector<float> total_area;
  total_area.resize(Ncells,0.f);
  for( std::size_t p=0; p<N; p++ ){
    if( prim_vol[p]>=0 ){
      uint gridcell = prim_vol[p];
      total_area.at(gridcell) += context->getPrimitiveArea(UUIDs_all.at(p));
      context->setPrimitiveData(UUIDs_all.at(p),"gridCell",helios::HELIOS_TYPE_UINT,1,&gridcell);
    }
  }

  std::vector<float> output_LeafArea;
  output_LeafArea.resize(Ncells,0.f);

  for( int v=0; v<Ncells; v++ ){
    output_LeafArea.at(v) =  total_area.at(v);
    if( context->doesPrimitiveDataExist(UUIDs_all.at(v), "gridCell") ){
      context->setPrimitiveData(UUIDs_all.at(v),"synthetic_leaf_area",total_area.at(v));
    }
  }
  
  free(prim_vol);

  CUDA_CHECK_ERROR( hipFree(d_prim_vol) );
  CUDA_CHECK_ERROR( hipFree(d_prim_xyz) );
  CUDA_CHECK_ERROR( hipFree(d_grid_center) );
  CUDA_CHECK_ERROR( hipFree(d_grid_anchor) );
  CUDA_CHECK_ERROR( hipFree(d_grid_size) );
  CUDA_CHECK_ERROR( hipFree(d_grid_rotation) );

  return output_LeafArea;

}

__global__ void LIDAR_CUDA::intersectPatches( const size_t Npulse, const int rays_per_pulse, const float3 origin, float3* d_raydir, const float exit_diameter, const float beam_divergence,const int Npatches, float3* d_patch_vertex, int* d_patch_textureID, const int Ntextures, int2* d_masksize, int2 masksize_max, bool* d_maskdata, float2* d_patch_uv, float* d_hit_t, float* d_hit_fnorm, int* d_hit_ID ){

  int idx = blockIdx.x*blockDim.x+threadIdx.x;

  int subpulse = blockIdx.y*blockDim.y+threadIdx.y;

  if( idx>=Npulse || subpulse>=rays_per_pulse ){
    return;
  }

  int i_global = idx*rays_per_pulse+subpulse;

  uint seed = tea<16>(subpulse,i_global);

  float2 disk_pt = d_sampleDisk( 0.5*exit_diameter, seed );
  
  float3 raydir = d_sphere2cart( 1.f, 0.5*M_PI-0.5*beam_divergence*rnd(seed), 2.f*M_PI*rnd(seed) );

  float zenith = acos_safe( d_raydir[idx].z/sqrt(d_raydir[idx].x*d_raydir[idx].x+d_raydir[idx].y*d_raydir[idx].y+d_raydir[idx].z*d_raydir[idx].z) );
  float azimuth = atan2_2pi( d_raydir[idx].x, d_raydir[idx].y );
  raydir = d_rotatePoint( raydir, zenith, 0.5*M_PI-azimuth );

  float3 disk_pt3 = d_rotatePoint( make_float3(disk_pt.x,disk_pt.y,0), zenith, azimuth );

  float tmin = d_hit_t[i_global];
  float nmin = d_hit_fnorm[i_global];
  for( int p=0; p<Npatches; p++ ){

    float3 v0 = d_patch_vertex[4*p];
    float3 v1 = d_patch_vertex[4*p+1];
    float3 v2 = d_patch_vertex[4*p+2];
    float3 v3 = d_patch_vertex[4*p+3];

    float3 anchor = v0;
    float3 normal = normalize( cross( v1-v0, v2-v0 ) );

    float3 a = v1-v0;
    float3 b = v3-v0;
    
    float t = ((anchor - origin - disk_pt3)*normal) / (raydir*normal); 
    
    if( t==t && t>1e-8 && t<tmin ){
			
      float3 point = origin + disk_pt3 + raydir * t;
      float3 d = point - anchor;
      
      float ddota = d*a;
      
      if (ddota > 0.0 && ddota < a*a ){
	
	float ddotb = d*b;

	if (ddotb > 0.0 && ddotb < b*b ){

	  if( d_patch_textureID[p]<0 ){//no texture mask
	    
	    tmin = t;
	    nmin = fabs(normal*raydir);
	    d_hit_ID[i_global] = p; 

	  }else{ //has texture mask

	    float amag = magnitude(a);
	    float bmag = magnitude(b);
	    float2 uv = make_float2( 1.f-ddota/amag/amag, ddotb/bmag/bmag );
	    int ID = d_patch_textureID[p];
	    int2 sz = d_masksize[ID];

	    float2 uvmin = d_patch_uv[2*p];
	    float2 duv;
	    duv.x = d_patch_uv[ 2*p+1 ].x - d_patch_uv[ 2*p ].x;
	    duv.y = d_patch_uv[ 2*p+1 ].y - d_patch_uv[ 2*p ].y;
	    int2 ind = make_int2( roundf(float(sz.x-1)*(uvmin.x+fabs(uv.x)*duv.x)), roundf(float(sz.y-1)*(uvmin.y+fabs(uv.y)*duv.y)) );
	    assert( ind.x>=0 && ind.x<sz.x && ind.y>=0 && ind.y<sz.y );

	    if( d_maskdata[ ID*masksize_max.x*masksize_max.y + ind.y*masksize_max.x + ind.x ] ){

	      tmin = t;
	      nmin = fabs(normal*raydir);
	      d_hit_ID[i_global] = p; 
	    
	    }
	    
	  }
	    
	}		
	
      }
    }
  }

  d_hit_t[i_global] = tmin;

  d_hit_fnorm[i_global] = nmin;
  
}

__global__ void LIDAR_CUDA::intersectTriangles( const size_t Npulse, const int rays_per_pulse, const float3 origin, float3* d_raydir, const float exit_diameter, const float beam_divergence, const int Ntriangles, const int Npatches, float3* d_tri_vertex, int* d_tri_textureID, const int Ntextures, int2* d_masksize, const int2 masksize_max, bool* d_maskdata, float2* d_tri_uv, float* d_hit_t, float* d_hit_fnorm, int* d_hit_ID ){

  int idx = blockIdx.x*blockDim.x+threadIdx.x;

  int subpulse = blockIdx.y*blockDim.y+threadIdx.y;

  if( idx>=Npulse || subpulse>=rays_per_pulse ){
    return;
  }

  int i_global = idx*rays_per_pulse+subpulse;

  uint seed = tea<16>(subpulse,i_global);

  float2 disk_pt = d_sampleDisk( 0.5*exit_diameter, seed );
  
  float3 raydir = d_sphere2cart( 1.f, 0.5*M_PI-0.5*beam_divergence*rnd(seed), 2.f*M_PI*rnd(seed) );

  float zenith = acos_safe( d_raydir[idx].z/sqrt(d_raydir[idx].x*d_raydir[idx].x+d_raydir[idx].y*d_raydir[idx].y+d_raydir[idx].z*d_raydir[idx].z) );
  float azimuth = atan2_2pi( d_raydir[idx].x, d_raydir[idx].y );
  raydir = d_rotatePoint( raydir, zenith, 0.5*M_PI-azimuth );

  float3 disk_pt3 = d_rotatePoint( make_float3(disk_pt.x,disk_pt.y,0), zenith, azimuth );

  float tmin = d_hit_t[i_global];
  float nmin = d_hit_fnorm[i_global];
  for( int tri=0; tri<Ntriangles; tri++ ){

    float3 v0 = d_tri_vertex[3*tri];
    float3 v1 = d_tri_vertex[3*tri+1];
    float3 v2 = d_tri_vertex[3*tri+2];

    float a = v0.x - v1.x, b = v0.x - v2.x, c = raydir.x, d = v0.x - origin.x - disk_pt3.x; 
    float e = v0.y - v1.y, f = v0.y - v2.y, g = raydir.y, h = v0.y - origin.y - disk_pt3.y;
    float i = v0.z - v1.z, j = v0.z - v2.z, k = raydir.z, l = v0.z - origin.z - disk_pt3.z;
		
    float m = f * k - g * j, n = h * k - g * l, p = f * l - h * j;
    float q = g * i - e * k, s = e * j - f * i;
  
    float inv_denom  = 1.f / (a * m + b * q + c * s);
  
    float e1 = d * m - b * n - c * p;
    float beta = e1 * inv_denom;
  
    if (beta > 0.0){
      
      float r = r = e * l - h * i;
      float e2 = a * n + d * q + c * r;
      float gamma = e2 * inv_denom;
      
      if (gamma > 0.0 && beta + gamma < 1.0 ){
	
	float e3 = a * p - b * r + d * s;
	float t = e3 * inv_denom;

	if( t>1e-6 && t<tmin ){

	  if( d_tri_textureID[tri]<0 ){//no texture mask	  
	    tmin = t;
	    float3 normal = normalize( cross( v1-v0, v2-v0 ) );
	    nmin = fabs(normal*raydir);
	    d_hit_ID[i_global] = Npatches+tri; 
	  }else{//has texture mask

	  int ID = d_tri_textureID[tri];
	  int2 sz = d_masksize[ID];
	  
	  float2 uv0 = d_tri_uv[tri*3];
	  float2 uv1 = d_tri_uv[tri*3+1];
	  float2 uv2 = d_tri_uv[tri*3+2];

	    float2 uv;
	    uv.x = (uv0.x + beta*(uv1.x-uv0.x) + gamma*(uv2.x-uv0.x));
	    uv.y = (uv0.y + beta*(uv1.y-uv0.y) + gamma*(uv2.y-uv0.y));

	    uint2 ind = make_uint2( roundf(float(sz.x-1)*fabs(uv.x)), roundf(float(sz.y-1)*fabs(uv.y)) );
	    assert( ind.x<sz.x && ind.y<sz.y );

	    if( d_maskdata[ ID*masksize_max.x*masksize_max.y + ind.y*masksize_max.x + ind.x ] ){
	      tmin = t;
	      float3 normal = normalize( cross( v1-v0, v2-v0 ) );
	      nmin = fabs(normal*raydir);
	      d_hit_ID[i_global] = Npatches+tri; 
	    }
	    
	    
	  }
	}
      
      }
    }
  }

  d_hit_t[i_global] = tmin;

  d_hit_fnorm[i_global] = nmin;
  
}
